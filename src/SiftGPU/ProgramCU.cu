
#include <hip/hip_runtime.h>
////////////////////////////////////////////////////////////////////////////
//	File:		ProgramCU.cu
//	Author:		Changchang Wu
//	Description : implementation of ProgramCU and all CUDA kernels
//
//	Copyright (c) 2007 University of North Carolina at Chapel Hill
//	All Rights Reserved
//
//	Permission to use, copy, modify and distribute this software and its
//	documentation for educational, research and non-profit purposes, without
//	fee, and without a written agreement is hereby granted, provided that the
//	above copyright notice and the following paragraph appear in all copies.
//	
//	The University of North Carolina at Chapel Hill make no representations
//	about the suitability of this software for any purpose. It is provided
//	'as is' without express or implied warranty. 
//
//	Please send BUG REPORTS to ccwu@cs.unc.edu
//
////////////////////////////////////////////////////////////////////////////

#if defined(CUDA_SIFTGPU_ENABLED)

#include "GL/glew.h"
#include "stdio.h"

#include <iostream>
#include <assert.h>
#include "CuTexImage.h"
#include "ProgramCU.h"
#include "GlobalUtil.h"

//----------------------------------------------------------------
//Begin SiftGPU setting section.
//////////////////////////////////////////////////////////
#define IMUL(X,Y) __mul24(X,Y)
//#define FDIV(X,Y) ((X)/(Y))
#define FDIV(X,Y) __fdividef(X,Y)

/////////////////////////////////////////////////////////
//filter kernel width range (don't change this)
#define KERNEL_MAX_WIDTH 33
#define KERNEL_MIN_WIDTH 5

//////////////////////////////////////////////////////////
//horizontal filter block size (32, 64, 128, 256, 512)
#define FILTERH_TILE_WIDTH 128
//thread block for vertical filter. FILTERV_BLOCK_WIDTH can be (4, 8 or 16)
#define FILTERV_BLOCK_WIDTH 16
#define FILTERV_BLOCK_HEIGHT 32
//The corresponding image patch for a thread block
#define FILTERV_PIXEL_PER_THREAD 4
#define FILTERV_TILE_WIDTH FILTERV_BLOCK_WIDTH
#define FILTERV_TILE_HEIGHT (FILTERV_PIXEL_PER_THREAD * FILTERV_BLOCK_HEIGHT)

//////////////////////////////////////////////////////////
//thread block size for computing Difference of Gaussian
#define DOG_BLOCK_LOG_DIMX 7
#define DOG_BLOCK_LOG_DIMY 0
#define DOG_BLOCK_DIMX (1 << DOG_BLOCK_LOG_DIMX)
#define DOG_BLOCK_DIMY (1 << DOG_BLOCK_LOG_DIMY)

//////////////////////////////////////////////////////////
//thread block size for keypoint detection
#define KEY_BLOCK_LOG_DIMX 5  // 3
#define KEY_BLOCK_LOG_DIMY 2  // 3
#define KEY_BLOCK_DIMX (1<<KEY_BLOCK_LOG_DIMX)
#define KEY_BLOCK_DIMY (1<<KEY_BLOCK_LOG_DIMY)
//make KEY_BLOCK_LOG_DIMX 4 will make the write coalesced..
//but it seems uncoalesced writes don't affect the speed

//////////////////////////////////////////////////////////
//thread block size for initializing list generation (64, 128, 256, 512 ...)
#define HIST_INIT_WIDTH 128
//thread block size for generating feature list (32, 64, 128, 256, 512, ...)
#define LISTGEN_BLOCK_DIM 128

/////////////////////////////////////////////////////////
//how many keypoint orientations to compute in a block
#define ORIENTATION_COMPUTE_PER_BLOCK 64
//how many keypoint descriptor to compute in a block (2, 4, 8, 16, 32)
#define DESCRIPTOR_COMPUTE_PER_BLOCK	4
#define DESCRIPTOR_COMPUTE_BLOCK_SIZE	(16 * DESCRIPTOR_COMPUTE_PER_BLOCK)

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  // block size for the keypoint descriptor normalization kernel 
  // it is assumed that one descriptor is processed by one warp 
  // -> have to be multiple of warp size (32)
  #define DESCRIPTOR_NORMALIZE_PER_BLOCK	128
#else
  //how many keypoint descriptor to normalized in a block (32, ...)
  #define DESCRIPTOR_NORMALIZE_PER_BLOCK	32
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

///////////////////////////////////////////
//Thread block size for visualization 
//(This doesn't affect the speed of computation)
#define BLOCK_LOG_DIM 4
#define BLOCK_DIM (1 << BLOCK_LOG_DIM)

//End SiftGPU setting section.
//----------------------------------------------------------------

__device__ __constant__ float d_kernel[KERNEL_MAX_WIDTH];

texture<float, 1, cudaReadModeElementType> texData;
texture<unsigned char, 1, cudaReadModeNormalizedFloat> texDataB;
texture<float2, 2, cudaReadModeElementType> texDataF2;
texture<float4, 1, cudaReadModeElementType> texDataF4;
texture<int4, 1, cudaReadModeElementType> texDataI4;
texture<int4, 1, cudaReadModeElementType> texDataList;

//template<int i>	 __device__ float Conv(float *data)		{    return Conv<i-1>(data) + data[i]*d_kernel[i];}
//template<>		__device__ float Conv<0>(float *data)	{    return data[0] * d_kernel[0];					}
  
//////////////////////////////////////////////////////////////
template<int FW> __global__ void FilterH( float* d_result, int width)
{
  const int HALF_WIDTH = FW >> 1;
  const int CACHE_WIDTH = FILTERH_TILE_WIDTH + FW -1;
  const int CACHE_COUNT = 2 + (CACHE_WIDTH - 2)/ FILTERH_TILE_WIDTH;

  __shared__ float data[CACHE_WIDTH];

  const int bcol = IMUL(blockIdx.x, FILTERH_TILE_WIDTH);
  const int col =  bcol + threadIdx.x;
  const int index_min = IMUL(blockIdx.y, width);
  const int index_max = index_min + width - 1;
  int src_index = index_min + bcol - HALF_WIDTH + threadIdx.x;
  int cache_index = threadIdx.x;
  float value = 0;

#pragma unroll
  for(int j = 0; j < CACHE_COUNT; ++j)
  {
    if(cache_index < CACHE_WIDTH)
    {
      int fetch_index = src_index < index_min? index_min : (src_index > index_max ? index_max : src_index);
      data[cache_index] = tex1Dfetch(texData,fetch_index);
      src_index += FILTERH_TILE_WIDTH;
      cache_index += FILTERH_TILE_WIDTH;
    }
  }

  __syncthreads(); 
  if(col >= width)
    return;

#pragma unroll
  for(int i = 0; i < FW; ++i)
  {
    value += (data[threadIdx.x + i]* d_kernel[i]);
  }
  // value = Conv<FW-1>(data + threadIdx.x);
  d_result[index_min + col] = value;
}

////////////////////////////////////////////////////////////////////
template<int  FW>  __global__ void FilterV(float* d_result, int width, int height)
{
  const int HALF_WIDTH = FW >> 1;
  const int CACHE_WIDTH = FW + FILTERV_TILE_HEIGHT - 1;
  const int TEMP = CACHE_WIDTH & 0xf;

  // add some extra space to avoid bank conflict
#if FILTERV_TILE_WIDTH == 16
  // make the stride 16 * n +/- 1
  const int EXTRA = (TEMP == 1 || TEMP == 0) ? 1 - TEMP : 15 - TEMP;
#elif FILTERV_TILE_WIDTH == 8
  // make the stride 16 * n +/- 2
  const int EXTRA = (TEMP == 2 || TEMP == 1 || TEMP == 0) ? 2 - TEMP : (TEMP == 15? 3 : 14 - TEMP);
#elif FILTERV_TILE_WIDTH == 4
  // make the stride 16 * n +/- 4
  const int EXTRA = (TEMP >=0 && TEMP <=4) ? 4 - TEMP : (TEMP > 12? 20 - TEMP : 12 - TEMP);
#else
#error
#endif

  const int CACHE_TRUE_WIDTH = CACHE_WIDTH + EXTRA;
  const int CACHE_COUNT = (CACHE_WIDTH + FILTERV_BLOCK_HEIGHT - 1) / FILTERV_BLOCK_HEIGHT;
  const int WRITE_COUNT = (FILTERV_TILE_HEIGHT + FILTERV_BLOCK_HEIGHT -1) / FILTERV_BLOCK_HEIGHT;

  __shared__ float data[CACHE_TRUE_WIDTH * FILTERV_TILE_WIDTH];

  const int row_block_first = IMUL(blockIdx.y, FILTERV_TILE_HEIGHT);
  const int col = IMUL(blockIdx.x, FILTERV_TILE_WIDTH) + threadIdx.x;
  const int row_first = row_block_first - HALF_WIDTH;
  const int data_index_max = IMUL(height - 1, width) + col;
  const int cache_col_start = threadIdx.y;	
  const int cache_row_start = IMUL(threadIdx.x, CACHE_TRUE_WIDTH);
  int cache_index = cache_col_start + cache_row_start;
  int data_index = IMUL(row_first + cache_col_start, width) + col;

  if(col < width) 
  {
#pragma unroll
    for(int i = 0; i < CACHE_COUNT; ++i)
    {
      if(cache_col_start < CACHE_WIDTH - i * FILTERV_BLOCK_HEIGHT) 
      {
        int fetch_index = data_index < col ? col : (data_index > data_index_max? data_index_max : data_index);
        data[cache_index + i * FILTERV_BLOCK_HEIGHT] = tex1Dfetch(texData,fetch_index);
        data_index += IMUL(FILTERV_BLOCK_HEIGHT, width);
      }
    }
  }
  __syncthreads();
	
  if(col >= width)
    return;

  int row = row_block_first + threadIdx.y;
  int index_start = cache_row_start + threadIdx.y;

#pragma unroll
  for(int i = 0; i < WRITE_COUNT; ++i, row += FILTERV_BLOCK_HEIGHT, index_start += FILTERV_BLOCK_HEIGHT)
  {
    if(row < height)
    {
      int index_dest = IMUL(row, width) + col;
      float value = 0;

#pragma unroll
      for(int i = 0; i < FW; ++i)
      {
        value += (data[index_start + i] * d_kernel[i]);
      }
      d_result[index_dest] = value;
    }
  }
}

template<int LOG_SCALE> __global__ void UpsampleKernel(float* d_result, int width)
{
  const int SCALE = (1 << LOG_SCALE), SCALE_MASK = (SCALE - 1);
  const float INV_SCALE = 1.0f / (float(SCALE));
  int col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;

  if(col >= width)
    return;

  int row = blockIdx.y >> LOG_SCALE;
  int index = row * width + col;
  int dst_row = blockIdx.y;
  int dst_idx = (width * dst_row + col) * SCALE;
  int helper = blockIdx.y & SCALE_MASK;

  if (helper)
  {
    float v11 = tex1Dfetch(texData, index);
    float v12 = tex1Dfetch(texData, index + 1);
    index += width;
    float v21 = tex1Dfetch(texData, index);
    float v22 = tex1Dfetch(texData, index + 1);

    float w1 = INV_SCALE * helper, w2 = 1.0 - w1;
    float v1 = (v21 * w1  + w2 * v11);
    float v2 = (v22 * w1  + w2 * v12);

    d_result[dst_idx] = v1;

#pragma unroll
    for(int i = 1; i < SCALE; ++i)
    {
      const float r2 = i * INV_SCALE;
      const float r1 = 1.0f - r2; 
      d_result[dst_idx +i] = v1 * r1 + v2 * r2;
    }
  }
  else
  {
    float v1 = tex1Dfetch(texData, index);
    float v2 = tex1Dfetch(texData, index + 1);

    d_result[dst_idx] = v1;

#pragma unroll
    for(int i = 1; i < SCALE; ++i)
    {
      const float r2 = i * INV_SCALE;
      const float r1 = 1.0f - r2; 
      d_result[dst_idx +i] = v1 * r1 + v2 * r2;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////////////
void ProgramCU::SampleImageU(CuTexImage *dst, CuTexImage *src, int log_scale)
{
  int width = src->GetImgWidth(), height = src->GetImgHeight();
  src->BindTexture(texData);

  dim3 grid((width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, height << log_scale);
  dim3 block(FILTERH_TILE_WIDTH);

  switch(log_scale)
  {
    case 1:
      UpsampleKernel<1> <<< grid, block>>> ((float*) dst->_cuData, width);
      break;
    case 2:
      UpsampleKernel<2> <<< grid, block>>> ((float*) dst->_cuData, width);
      break;
    case 3:
      UpsampleKernel<3> <<< grid, block>>> ((float*) dst->_cuData, width);
      break;
    default:
      break;
  }
}

template<int LOG_SCALE> __global__ void DownsampleKernel(float* d_result, int src_width, int dst_width)
{
  const int dst_col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;

  if(dst_col >= dst_width)
    return;

  const int src_col = min((dst_col << LOG_SCALE), (src_width - 1));
  const int dst_row = blockIdx.y; 
  const int src_row = blockIdx.y << LOG_SCALE;
  const int src_idx = IMUL(src_row, src_width) + src_col;
  const int dst_idx = IMUL(dst_width, dst_row) + dst_col;

  d_result[dst_idx] = tex1Dfetch(texData, src_idx);
}

__global__ void DownsampleKernel(float* d_result, int src_width, int dst_width, const int log_scale)
{
  const int dst_col = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;

  if(dst_col >= dst_width)
    return;

  const int src_col = min((dst_col << log_scale), (src_width - 1));
  const int dst_row = blockIdx.y; 
  const int src_row = blockIdx.y << log_scale;
  const int src_idx = IMUL(src_row, src_width) + src_col;
  const int dst_idx = IMUL(dst_width, dst_row) + dst_col;

  d_result[dst_idx] = tex1Dfetch(texData, src_idx);
}

void ProgramCU::SampleImageD(CuTexImage *dst, CuTexImage *src, int log_scale)
{
  int src_width = src->GetImgWidth();
  int dst_width = dst->GetImgWidth();

  src->BindTexture(texData);
  dim3 grid((dst_width +  FILTERH_TILE_WIDTH - 1)/ FILTERH_TILE_WIDTH, dst->GetImgHeight());
  dim3 block(FILTERH_TILE_WIDTH);

  switch(log_scale)
  {
    case 1:
      DownsampleKernel<1> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);
      break;
    case 2:
      DownsampleKernel<2> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);
      break;
    case 3:
      DownsampleKernel<3> <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width);
      break;
    default:
      DownsampleKernel    <<< grid, block>>> ((float*) dst->_cuData, src_width, dst_width, log_scale);
  }
}

__global__ void ChannelReduce_Kernel(float* d_result)
{
  int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;

  d_result[index] = tex1Dfetch(texData, index*4);
}

__global__ void ChannelReduce_Convert_Kernel(float* d_result)
{
  int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;
  float4 rgba = tex1Dfetch(texDataF4, index);

  d_result[index] = 0.299f * rgba.x + 0.587f* rgba.y + 0.114f * rgba.z;
}

void ProgramCU::ReduceToSingleChannel(CuTexImage* dst, CuTexImage* src, int convert_rgb)
{
  int width = src->GetImgWidth();
  int height = dst->GetImgHeight() ;

  dim3 grid((width * height +  FILTERH_TILE_WIDTH - 1) / FILTERH_TILE_WIDTH);
  dim3 block(FILTERH_TILE_WIDTH);

  if(convert_rgb)
  {
    src->BindTexture(texDataF4);
    ChannelReduce_Convert_Kernel<<<grid, block>>>((float*)dst->_cuData);
  }
  else
  {
    src->BindTexture(texData);
    ChannelReduce_Kernel<<<grid, block>>>((float*)dst->_cuData);
  }
}

__global__ void ConvertByteToFloat_Kernel(float* d_result)
{
  int index = IMUL(blockIdx.x, FILTERH_TILE_WIDTH) + threadIdx.x;

  d_result[index] = tex1Dfetch(texDataB, index);
}

void ProgramCU::ConvertByteToFloat(CuTexImage*src, CuTexImage* dst)
{
  int width = src->GetImgWidth();
  int height = dst->GetImgHeight() ;
  dim3 grid((width * height +  FILTERH_TILE_WIDTH - 1) / FILTERH_TILE_WIDTH);
  dim3 block(FILTERH_TILE_WIDTH);

  src->BindTexture(texDataB);

  ConvertByteToFloat_Kernel<<<grid, block>>>((float*)dst->_cuData);
}

void ProgramCU::CreateFilterKernel(float sigma, float* kernel, int& width)
{
  int i, sz = int( ceil( GlobalUtil::_FilterWidthFactor * sigma -0.5) ); //
  width = 2*sz + 1;

  if(width > KERNEL_MAX_WIDTH)
  {
    //filter size truncation
    sz = KERNEL_MAX_WIDTH >> 1;
    width = KERNEL_MAX_WIDTH;
  }
  else if(width < KERNEL_MIN_WIDTH)
  {
    sz = KERNEL_MIN_WIDTH >> 1;
    width =KERNEL_MIN_WIDTH;
  }

  float rv = 1.0f / (sigma*sigma), v, ksum = 0; 

  // pre-compute filter
  for( i = -sz ; i <= sz ; ++i) 
  {
    kernel[i+sz] =  v = exp(-0.5f * i * i *rv) ;
    ksum += v;
  }

  //normalize the kernel
  rv = 1.0f / ksum;
  for(i = 0; i< width ;i++)
    kernel[i] *= rv;
}

template<int FW> void ProgramCU::FilterImage(CuTexImage *dst, CuTexImage *src, CuTexImage* buf)
{
  int width = src->GetImgWidth();
  int height = src->GetImgHeight();

  //horizontal filtering
  src->BindTexture(texData);

  dim3 gridh((width +  FILTERH_TILE_WIDTH - 1)/FILTERH_TILE_WIDTH, height);
  dim3 blockh(FILTERH_TILE_WIDTH);

  FilterH<FW><<<gridh, blockh>>>((float*)buf->_cuData, width);

  CheckErrorCUDA("FilterH");

  ///vertical filtering
  buf->BindTexture(texData);

  dim3 gridv((width + FILTERV_TILE_WIDTH - 1)/FILTERV_TILE_WIDTH, (height + FILTERV_TILE_HEIGHT - 1)/FILTERV_TILE_HEIGHT);
  dim3 blockv(FILTERV_TILE_WIDTH, FILTERV_BLOCK_HEIGHT);

  FilterV<FW><<<gridv, blockv>>>((float*)dst->_cuData, width, height); 

  CheckErrorCUDA("FilterV");
}

//////////////////////////////////////////////////////////////////////
// tested on 2048x1500 image, the time on pyramid construction is
// OpenGL version : 18ms
// CUDA version: 28 ms
void ProgramCU::FilterImage(CuTexImage *dst, CuTexImage *src, CuTexImage* buf, float sigma)
{
  float filter_kernel[KERNEL_MAX_WIDTH];
  int width;

  CreateFilterKernel(sigma, filter_kernel, width);
  cudaMemcpyToSymbol(d_kernel, filter_kernel, width * sizeof(float), 0, cudaMemcpyHostToDevice);

  switch(width)
  {
    case 5:		FilterImage< 5>(dst, src, buf);	break;
    case 7:		FilterImage< 7>(dst, src, buf);	break;
    case 9:		FilterImage< 9>(dst, src, buf);	break;
    case 11:	FilterImage<11>(dst, src, buf);	break;
    case 13:	FilterImage<13>(dst, src, buf);	break;
    case 15:	FilterImage<15>(dst, src, buf);	break;
    case 17:	FilterImage<17>(dst, src, buf);	break;
    case 19:	FilterImage<19>(dst, src, buf);	break;
    case 21:	FilterImage<21>(dst, src, buf);	break;
    case 23:	FilterImage<23>(dst, src, buf);	break;
    case 25:	FilterImage<25>(dst, src, buf);	break;
    case 27:	FilterImage<27>(dst, src, buf);	break;
    case 29:	FilterImage<29>(dst, src, buf);	break;
    case 31:	FilterImage<31>(dst, src, buf);	break;
    case 33:	FilterImage<33>(dst, src, buf);	break;
    default:	break;
  }
}

texture<float, 1, cudaReadModeElementType> texC;
texture<float, 1, cudaReadModeElementType> texP;
texture<float, 1, cudaReadModeElementType> texN;

#ifdef GPU_HESSIAN

texture<float, 1, cudaReadModeElementType> texG;

// compute 3x3 Hessian values from symmetric differences
#define COMPUTE_HESSIAN(tex, idx)                 \
  float v11 = tex1Dfetch(tex, idx - width - 1);   \
  float v12 = tex1Dfetch(tex, idx - width);       \
  float v13 = tex1Dfetch(tex, idx - width + 1);   \
                                                  \
  float v21 = tex1Dfetch(tex, idx - 1);           \
  float v22 = tex1Dfetch(tex, idx);               \
  float v23 = tex1Dfetch(tex, idx + 1);           \
                                                  \
  float v31 = tex1Dfetch(tex, idx + width - 1);   \
  float v32 = tex1Dfetch(tex, idx + width);       \
  float v33 = tex1Dfetch(tex, idx + width + 1);   \
                                                  \
  float Lxx = (v21 - 2.0f*v22 + v23);             \
  float Lyy = (v12 - 2.0f*v22 + v32);             \
  float Lxy = (v13 - v11 + v31 - v33) * 0.25f;    \


void __global__ ComputeHessian_Kernel(float *hessian, float2 *got, int width, int height, float norm)
{
  int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
  int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;

  if((col < width) && (row < height)) 
  {
    int index = IMUL(row, width) + col;

    COMPUTE_HESSIAN(texC, index)

    // compute determinant of hessian matrix, normalize and write out
    hessian[index] = (Lxx*Lyy - Lxy*Lxy)*norm;

    // precompute gradient and rotation
    float dx = v23 - v21;
    float dy = v32 - v12;
    float gradient = 0.5f * sqrt(dx*dx  + dy*dy);
    float rot = ((gradient == 0.0f) ? 0.0f : atan2(dy, dx));

    got[index] = make_float2(gradient, rot);
  }
}

void __global__ ComputeHessian_Kernel(float *hessian, int width, int height, float norm)
{
  int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
  int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;

  if((col < width) && (row < height)) 
  {
    int index = IMUL(row, width) + col;

    COMPUTE_HESSIAN(texC, index)

    // compute determinant of hessian matrix, normalize and write out
    hessian[index] = (Lxx*Lyy - Lxy*Lxy)*norm;
  }
}

void ProgramCU::ComputeHessian(CuTexImage* gus, CuTexImage* dog, CuTexImage* got, float norm)
{
  int width = gus->GetImgWidth();
  int height = gus->GetImgHeight();

  dim3 grid((width + DOG_BLOCK_DIMX - 1) / DOG_BLOCK_DIMX, (height + DOG_BLOCK_DIMY - 1) / DOG_BLOCK_DIMY);
  dim3 block(DOG_BLOCK_DIMX, DOG_BLOCK_DIMY);

  gus->BindTexture(texC);

  if(got->_cuData)
    ComputeHessian_Kernel<<<grid, block>>>((float*)dog->_cuData, (float2*)got->_cuData, width, height, norm*norm);
  else
    ComputeHessian_Kernel<<<grid, block>>>((float*)dog->_cuData, width, height, norm*norm);
}

#else

void __global__ ComputeDOG_Kernel(float* d_dog, float2* d_got, int width, int height)
{
  int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
  int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;

  if(col < width && row < height) 
  {
    int index = IMUL(row, width) + col;
    float vp = tex1Dfetch(texP, index);
    float v = tex1Dfetch(texC, index);

    d_dog[index] = v - vp;

    float vxn = tex1Dfetch(texC, index + 1);
    float vxp = tex1Dfetch(texC, index - 1);
    float vyp = tex1Dfetch(texC, index - width);
    float vyn = tex1Dfetch(texC, index + width);
    float dx = vxn - vxp;
    float dy = vyn - vyp;
    float grd = 0.5f * sqrt(dx * dx  + dy * dy);
    float rot = (grd == 0.0f ? 0.0f : atan2(dy, dx));

    d_got[index] = make_float2(grd, rot);
  }
}

void __global__ ComputeDOG_Kernel(float* d_dog, int width, int height)
{
  int row = (blockIdx.y << DOG_BLOCK_LOG_DIMY) + threadIdx.y;
  int col = (blockIdx.x << DOG_BLOCK_LOG_DIMX) + threadIdx.x;

  if(col < width && row < height) 
  {
    int index = IMUL(row, width) + col;
    float vp = tex1Dfetch(texP, index);
    float v = tex1Dfetch(texC, index);
    d_dog[index] = v - vp;
  }
}

void ProgramCU::ComputeDOG(CuTexImage* gus, CuTexImage* dog, CuTexImage* got)
{
  int width = gus->GetImgWidth();
  int height = gus->GetImgHeight();
  dim3 grid((width + DOG_BLOCK_DIMX - 1)/ DOG_BLOCK_DIMX,  (height + DOG_BLOCK_DIMY - 1)/DOG_BLOCK_DIMY);
  dim3 block(DOG_BLOCK_DIMX, DOG_BLOCK_DIMY);

  gus->BindTexture(texC);
  (gus -1)->BindTexture(texP);

  if(got->_cuData)
    ComputeDOG_Kernel<<<grid, block>>>((float*) dog->_cuData, (float2*) got->_cuData, width, height);
  else
    ComputeDOG_Kernel<<<grid, block>>>((float*) dog->_cuData, width, height);
}

#endif // GPU_HESSIAN

#ifdef GPU_HESSIAN
  // GPU_HESSIAN: added test (response<0) and (response>0)
  #define READ_CMP_DOG_DATA(datai, tex, idx)  \
    datai[0] = tex1Dfetch(tex, idx - 1);      \
    datai[1] = tex1Dfetch(tex, idx);          \
    datai[2] = tex1Dfetch(tex, idx + 1);      \
    if(response > nmax)                       \
    {                                         \
      nmax = max(nmax, datai[0]);             \
      nmax = max(nmax, datai[1]);             \
      nmax = max(nmax, datai[2]);             \
      if((response < nmax) || (response < 0)) \
        goto key_finish;                      \
    }                                         \
    else                                      \
    {                                         \
      nmin = min(nmin, datai[0]);             \
      nmin = min(nmin, datai[1]);             \
      nmin = min(nmin, datai[2]);             \
      if((response > nmin) || (response > 0)) \
        goto key_finish;                      \
    }
#else
  #define READ_CMP_DOG_DATA(datai, tex, idx)  \
    datai[0] = tex1Dfetch(tex, idx - 1);      \
    datai[1] = tex1Dfetch(tex, idx);          \
    datai[2] = tex1Dfetch(tex, idx + 1);      \
    if(response > nmax)                       \
    {                                         \
      nmax = max(nmax, datai[0]);             \
      nmax = max(nmax, datai[1]);             \
      nmax = max(nmax, datai[2]);             \
      if(response < nmax)                     \
        goto key_finish;                      \
    }                                         \
    else                                      \
    {                                         \
      nmin = min(nmin, datai[0]);             \
      nmin = min(nmin, datai[1]);             \
      nmin = min(nmin, datai[2]);             \
      if(response > nmin)                     \
        goto key_finish;                      \
    }
#endif // GPU_HESSIAN

void __global__ ComputeKEY_Kernel(float4 *d_key, int width, int colmax, int rowmax, float dog_threshold0, float dog_threshold, float edge_threshold, int subpixel_localization
#if defined GENERATE_FEATURE_LIST_USING_ATOMICS && (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED)
		  , int *featureTexLen
#endif // GENERATE_FEATURE_LIST_USING_ATOMICS && (GPU_HESSIAN || GPU_SIFT_MODIFIED)
	)
{
  int row = (blockIdx.y << KEY_BLOCK_LOG_DIMY) + threadIdx.y;
  int col = (blockIdx.x << KEY_BLOCK_LOG_DIMX) + threadIdx.x;

  float data[3][3];
  float datap[3][3];
  float datan[3][3];
  float response = 0.0f;
  int index = IMUL(row, width) + col;
  int idx[3] = {index - width, index, index + width};
  float nmax, nmin, result = 0.0f;
  float dx = 0, dy = 0, ds = 0;
  int in_image = 0;
  bool offset_test_passed = true;
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  unsigned short pointType = FEATURE_TYPE_NONE;
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  if((row > 0) && (col > 0) && (row < rowmax) && (col < colmax))
  {
    in_image = 1;

    data[1][1] = response = tex1Dfetch(texC, idx[1]);
    if(fabs(response) <= dog_threshold0)
      goto key_finish;

    // fetch left and right neighbour
    data[1][0] = tex1Dfetch(texC, idx[1] - 1);
    data[1][2] = tex1Dfetch(texC, idx[1] + 1);
    nmax = max(data[1][0], data[1][2]);
    nmin = min(data[1][0], data[1][2]);

    if((response <= nmax) && (response >= nmin))
      goto key_finish;

    //if((response > nmax && response < 0 )|| (response < nmin && response > 0)) goto key_finish;
	// fetch values from the row above
    READ_CMP_DOG_DATA(data[0], texC, idx[0]);
	// fetch values from one the row below
    READ_CMP_DOG_DATA(data[2], texC, idx[2]);

    // edge supression
    float vx2 = response * 2.0f;
    float fxx = data[1][0] + data[1][2] - vx2;
    float fyy = data[0][1] + data[2][1] - vx2;
    float fxy = 0.25f * (data[2][2] + data[0][0] - data[2][0] - data[0][2]);
    float temp1 = fxx * fyy - fxy * fxy;
    float temp2 = (fxx + fyy) * (fxx + fyy);

    if((temp1 <= 0) || (temp2 > edge_threshold * temp1))
      goto key_finish; // local neighbourhood looks like an edge

    // read the previous level
    READ_CMP_DOG_DATA(datap[0], texP, idx[0]);
    READ_CMP_DOG_DATA(datap[1], texP, idx[1]);
    READ_CMP_DOG_DATA(datap[2], texP, idx[2]);

    // read the next level
    READ_CMP_DOG_DATA(datan[0], texN, idx[0]);
    READ_CMP_DOG_DATA(datan[1], texN, idx[1]);
    READ_CMP_DOG_DATA(datan[2], texN, idx[2]);

    if(subpixel_localization)
    {
      // subpixel localization
      float fx = 0.5f * (data[1][2] - data[1][0]);
      float fy = 0.5f * (data[2][1] - data[0][1]);
      float fs = 0.5f * (datan[1][1] - datap[1][1]);

      float fss = (datan[1][1] + datap[1][1] - vx2);
      float fxs = 0.25f * (datan[1][2] + datap[1][0] - datan[1][0] - datap[1][2]);
      float fys = 0.25f * (datan[2][1] + datap[0][1] - datan[0][1] - datap[2][1]);

      // need to solve dx, dy, ds;
      // |-fx|     | fxx fxy fxs |   |dx|
      // |-fy|  =  | fxy fyy fys | * |dy|
      // |-fs|     | fxs fys fss |   |ds|
      float4 A0 = (fxx > 0) ? make_float4(fxx, fxy, fxs, -fx) : make_float4(-fxx, -fxy, -fxs, fx);
      float4 A1 = (fxy > 0) ? make_float4(fxy, fyy, fys, -fy) : make_float4(-fxy, -fyy, -fys, fy);
      float4 A2 = (fxs > 0) ? make_float4(fxs, fys, fss, -fs) : make_float4(-fxs, -fys, -fss, fs);

      float maxa = max(max(A0.x, A1.x), A2.x);

      if(maxa >= 1e-10)
      {
        if(maxa == A1.x)
        {
          float4 TEMP = A1; A1 = A0; A0 = TEMP;
        }
        else if(maxa == A2.x)
        {
          float4 TEMP = A2; A2 = A0; A0 = TEMP;
        }
        A0.y /= A0.x;          A0.z /= A0.x;          A0.w /= A0.x;
        A1.y -= A1.x * A0.y;   A1.z -= A1.x * A0.z;   A1.w -= A1.x * A0.w;
        A2.y -= A2.x * A0.y;   A2.z -= A2.x * A0.z;   A2.w -= A2.x * A0.w;

        if(abs(A2.y) > abs(A1.y))
        {
          float4 TEMP = A2; A2 = A1; A1 = TEMP;
        }
        if(abs(A1.y) >= 1e-10) 
        {
          A1.z /= A1.y;          A1.w /= A1.y;
          A2.z -= A2.y * A1.z;   A2.w -= A2.y * A1.w;

          if(abs(A2.z) >= 1e-10) 
          {
            ds = A2.w / A2.z;
            dy = A1.w - ds * A1.z;
            dx = A0.w - ds * A0.z - dy * A0.y;

            response = data[1][1] + 0.5f * (dx * fx + dy * fy + ds * fs);

            offset_test_passed = (fabs(response) > dog_threshold) && (fabs(ds) < 1.0f) && (fabs(dx) < 1.0f) && (fabs(dy) < 1.0f);
          }
        }
      }
    }
 
    if(offset_test_passed)
#if defined GPU_HESSIAN
    {
      // find blob point type from Hessian matrix H, we know that:
      //   - if H is positive definite it is a DARK blob
      //   - if H is negative definite it is a BRIGHT blob
      //   - det H is negative it is a SADDLE point

      data[1][1] = tex1Dfetch(texG, idx[1]);
      data[1][0] = tex1Dfetch(texG, idx[1] - 1);
      data[1][2] = tex1Dfetch(texG, idx[1] + 1);

      if(response < 0)
      {
        pointType = FEATURE_TYPE_SADDLE_POINT;
      }
      else
      {
        // at this point we know that 2x2 determinant is positive
        // so only check the remaining 1x1 subdeterminant
        float Lxx = data[1][0] - 2*data[1][1] + data[1][2];

        pointType = (Lxx > 0) ? FEATURE_TYPE_DARK_BLOB : FEATURE_TYPE_BRIGHT_BLOB;
      }
    }
#elif defined GPU_SIFT_MODIFIED
      result = (response > nmax) ? FEATURE_TYPE_BRIGHT_BLOB : FEATURE_TYPE_DARK_BLOB;
#else
      result = (response > nmax) ? 1.0 : -1.0;
#endif // GPU_HESSIAN / GPU_SIFT_MODIFIED
  }

key_finish:

  if(in_image)
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  {
    // result: response 16b | 14b unused | 2b type
    unsigned int uspack = (((unsigned int)__float2half_rn(response)) << 16) | 0x00000004u | pointType;
	result = *((float *)(&uspack)); // __uint_as_float(uspack); // CUDA 7.5

    d_key[index] = make_float4(result, dx, dy, ds);
  }
#else
    d_key[index] = make_float4(result, dx, dy, ds);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

#if defined GENERATE_FEATURE_LIST_USING_ATOMICS && (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED)
  int count = __syncthreads_count(pointType != FEATURE_TYPE_NONE);

  if(threadIdx.x+threadIdx.y*blockDim.x == 0)
  {
	atomicAdd(featureTexLen, count);
  }
#endif // GENERATE_FEATURE_LIST_USING_ATOMICS && (GPU_HESSIAN || GPU_SIFT_MODIFIED)
}

void ProgramCU::ComputeKEY(CuTexImage* dog, CuTexImage* key
#if defined GPU_HESSIAN
    , CuTexImage* gus
#endif // GPU_HESSIAN
    , float Tdog, float Tedge
#if defined GENERATE_FEATURE_LIST_USING_ATOMICS && (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED)
    , int *featureTexLen, int featureTexIdx
#endif // GENERATE_FEATURE_LIST_USING_ATOMICS && (GPU_HESSIAN || GPU_SIFT_MODIFIED)
)
{
  int width = dog->GetImgWidth();
  int height = dog->GetImgHeight();

  float Tdog1 = (GlobalUtil::_SubpixelLocalization ? 0.8f : 1.0f) * Tdog;

  CuTexImage *dogp = dog - 1;
  CuTexImage *dogn = dog + 1;

  dim3 grid((width + KEY_BLOCK_DIMX - 1)/KEY_BLOCK_DIMX, (height + KEY_BLOCK_DIMY - 1)/KEY_BLOCK_DIMY);
  dim3 block(KEY_BLOCK_DIMX, KEY_BLOCK_DIMY);

  dogp->BindTexture(texP);
  dog->BindTexture(texC);
  dogn->BindTexture(texN);

#if defined GPU_HESSIAN
  gus->BindTexture(texG);
#endif // GPU_HESSIAN

  Tedge = (Tedge+1)*(Tedge+1) / Tedge;

  ComputeKEY_Kernel<<<grid, block>>>((float4*)key->_cuData, width, width-1, height-1, Tdog1, Tdog, Tedge, GlobalUtil::_SubpixelLocalization
#if defined GENERATE_FEATURE_LIST_USING_ATOMICS && (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED)
		  , featureTexLen + featureTexIdx
#endif // GENERATE_FEATURE_LIST_USING_ATOMICS && (GPU_HESSIAN || GPU_SIFT_MODIFIED)
	  );
}

#if defined GENERATE_FEATURE_LIST_USING_ATOMICS && (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED)

#define GENERATE_LIST_BLOCK_DIMX 32
#define GENERATE_LIST_BLOCK_DIMY 4

__device__ int GFL_WarpScan(int val, volatile int *sData, int threadID)
{
  // pad each warp with zeros
  // int idx = 2*threadIdx.x - (threadIdx.x & (warpSize-1)); // 1D
  // int id = threadIdx.x + threadIdx.y*blockDim.x; // 2D
  // int idx = 2*id - (id & (warpSize-1));
  int idx = 2*threadID - (threadID & (warpSize-1));

  sData[idx] = 0;
  idx += warpSize;

  int t = sData[idx] = val;

  sData[idx] = t = t + sData[idx -  1];
  sData[idx] = t = t + sData[idx -  2];
  sData[idx] = t = t + sData[idx -  4];
  sData[idx] = t = t + sData[idx -  8];
  sData[idx] = t = t + sData[idx - 16];

  return sData[idx-1];
}

__device__ unsigned int GFL_LaneMaskLt(int threadID)
{
  // const unsigned int lane = threadIdx.x & (warpSize-1);                            // 1D block
  // const unsigned int lane = (threadIdx.x + threadIdx.y*blockDim.x) & (warpSize-1); // 2D block
  const unsigned int lane = threadID & (warpSize-1);

  return (1 << (lane)) - 1;
}

__device__ unsigned int GFL_WarpPrefixSums(bool p, int threadID)
{
  const unsigned int mask = GFL_LaneMaskLt(threadID);
  unsigned int b = __ballot(p);

  return __popc(b & mask);
}

__device__ int GFL_BlockBinaryPrefixSums(int x, int idx)
{
  extern __shared__ int sData[];

  //int idx = threadIdx.x + threadIdx.y*blockDim.x;

  // A. Compute exclusive prefix sums within each warp
  int warpPrefix = GFL_WarpPrefixSums(x, idx);

  // int idx = threadIdx.x; // 1D
  // int idx = threadIdx.x + threadIdx.y*blockDim.x; // 2D
  int warpIdx = idx / warpSize;
  int laneIdx = idx & (warpSize - 1);

  // B. The last thread of each warp stores inclusive
  // prefix sum to the warp�s index in shared memory
  if(laneIdx == warpSize - 1)
    sData[warpIdx] = warpPrefix + x;

  __syncthreads();

  // C. One warp scans the warp partial sums
  if(idx < warpSize)
    sData[idx] = GFL_WarpScan(sData[idx], sData, idx);

  __syncthreads();

  // D. Each thread adds prefix sums of warp partial
  // sums to its own intra-warp prefix sums
  return warpPrefix + sData[warpIdx];
}

void __global__ ListGen_Kernel(int4* d_list, int len, int width, int height, int *counter)
{
  int row = IMUL(blockIdx.y, GENERATE_LIST_BLOCK_DIMY) + threadIdx.y;
  int col = IMUL(blockIdx.x, GENERATE_LIST_BLOCK_DIMX) + threadIdx.x;

  // read the detected keypoint type -> flag=0 (type == FEATURE_TYPE_NONE) / 1 (otherwise)
  unsigned int flag = 0;

  if((row > 0) && (col > 0) && (row < height-1) && (col < width-1))
  {
    int index = IMUL(row, width) + col;
    float4 value = tex1Dfetch(texDataF4, index);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    // value: (response 16b | 14b unused | 2b type), dx, dy, ds
    // type = *((unsigned int *)(&offset.x)) & 0x00000003u;
	flag = ((*((unsigned int *)(&value.x)) & 0x00000003u) != FEATURE_TYPE_NONE) ? 1 : 0;
#else
    // value: (response 16b | 16b type), dx, dy, ds ... type = -1, 0, +1
    unsigned int resultUInt = *((unsigned int*)(&value.x)) & 0x0000FFFFu; // float_as_uint(value.x) & 0x0000FFFFu; // CUDA 7.5
    float result = __half2float(resultUInt);

    flag = (fabs(result) > 0.5f) ? 1 : 0; 	// flag = (result != 0.0f) ? 1 : 0;
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
  }

  int idxWithinBlock = IMUL(threadIdx.y, blockDim.x) + threadIdx.x;

  // compute prefix sums for each thread int the block 
  int blockPrefixSum = GFL_BlockBinaryPrefixSums(flag, idxWithinBlock);

  // allocate enough space in the feature list to store all features in this block
  // number of features can be obtained by __syncthreads_count() or it is given
  // by the prefix sum of the last thread in the block plus one (if the last thread
  // represents detected feature)
  __shared__ int blockStart; // index in the feature list of the first keypoint in the block

  // int count = __syncthreads_count(flag);
  // if(idxWitninBlock == 0)
  // {
  //   blockStart = atomicAdd(counter, count);
  // }
  // __syncthreads();

  if(idxWithinBlock == IMUL(blockDim.y, blockDim.x) - 1)
  {
    blockStart = atomicAdd(counter, blockPrefixSum + flag);
  }
  __syncthreads();

  // put detected keypoint into the feature list
  if(flag)
    d_list[blockStart + blockPrefixSum] = make_int4(col, row, 0, 0);
}

#else

void __global__ InitHist_Kernel(int4* hist, int ws, int wd, int height)
{
  int row = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
  int col = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;

  if((row < height) && (col < wd))
  {
    int hidx = IMUL(row, wd) + col;
    int scol = col << 2;
    int sidx = IMUL(row, ws) + scol;
    int v[4] = {0, 0, 0, 0};

    // each thread process 4 subsequent colums values in the same row
    if((row > 0) && (row < height-1))
    {
#pragma unroll
      for(int i = 0; i < 4 ; ++i, ++scol)
      {
        float4 temp = tex1Dfetch(texDataF4, sidx+i);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
        unsigned int featureType = *((unsigned int*)(&temp.x)) & 0x00000003u; // float_as_uint(temp.x) & 0x00000003u; // CUDA 7.5

        v[i] = ((scol < ws-1) && (scol > 0) && (featureType != FEATURE_TYPE_NONE)) ? 1 : 0;
#else
        v[i] = ((scol < ws-1) && (scol > 0) && (temp.x != 0.0)) ? 1 : 0;
#endif // GPU_SIFT_MODIFIED || GPU_HESSIAN
      }
    }
    hist[hidx] = make_int4(v[0], v[1], v[2], v[3]);
  }
}

void ProgramCU::InitHistogram(CuTexImage* key, CuTexImage* hist)
{
  int ws = key->GetImgWidth();
  int hs = key->GetImgHeight();
  int wd = hist->GetImgWidth();
  int hd = hist->GetImgHeight();
  dim3 grid((wd + HIST_INIT_WIDTH - 1) / HIST_INIT_WIDTH, hd);
  dim3 block(HIST_INIT_WIDTH, 1);

  key->BindTexture(texDataF4);

  InitHist_Kernel<<<grid, block>>>((int4*) hist->_cuData, ws, wd, hd);
}

void __global__ ReduceHist_Kernel(int4* d_hist, int ws, int wd, int height)
{
  int row = IMUL(blockIdx.y, blockDim.y) + threadIdx.y;
  int col = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;

  if((row < height) && (col < wd))
  {
    int hidx = IMUL(row, wd) + col;
    int scol = col << 2;
    int sidx = IMUL(row, ws) + scol;
    int v[4] = {0, 0, 0, 0};

#pragma unroll
    for(int i = 0; (i < 4) && (scol < ws); ++i, ++scol)
    {
      int4 temp = tex1Dfetch(texDataI4, sidx + i);
      v[i] = temp.x + temp.y + temp.z + temp.w;
    }

    d_hist[hidx] = make_int4(v[0], v[1], v[2], v[3]);
  }
}

void ProgramCU::ReduceHistogram(CuTexImage* hist1, CuTexImage* hist2)
{
  int ws = hist1->GetImgWidth();
  int hs = hist1->GetImgHeight();
  int wd = hist2->GetImgWidth();
  int hd = hist2->GetImgHeight();
  int temp = (int)floor(logf(float(wd * 2 / 3)) / logf(2.0f));
  const int wi = min(7, max(temp , 0));

  hist1->BindTexture(texDataI4);

  const int BW = 1 << wi;
  const int BH = 1 << (7 - wi);
  dim3 grid((wd  + BW - 1) / BW,  (hd + BH -1) / BH);
  dim3 block(BW, BH);

  ReduceHist_Kernel<<<grid, block>>>((int4*)hist2->_cuData, ws, wd, hd);
}

void __global__ ListGen_Kernel(int4* d_list, int len, int width)
{
  int idx1 = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  // GPU_HESSIAN fix
  if(idx1 >= len)
    return;
#endif // GPU_HESSIAN

  int4 pos = tex1Dfetch(texDataList, idx1);

  int idx2 = IMUL(pos.y, width) + pos.x;
  int4 temp = tex1Dfetch(texDataI4, idx2);

  int  sum1 = temp.x + temp.y;
  int  sum2 = sum1 + temp.z;

  pos.x <<= 2;
  if(pos.z >= sum2)
  {
    pos.x += 3;
    pos.z -= sum2;
  }
  else if(pos.z >= sum1)
  {
    pos.x += 2;
    pos.z -= sum1;
  }
  else if(pos.z >= temp.x)
  {
    pos.x += 1;
    pos.z -= temp.x;
  }

  d_list[idx1] = pos;
}

#endif // GENERATE_FEATURE_LIST_USING_ATOMICS && (GPU_HESSIAN || GPU_SIFT_MODIFIED)

#if defined GENERATE_FEATURE_LIST_USING_ATOMICS && (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED)

// input list (x, y) (x, y) ....
void ProgramCU::GenerateList(CuTexImage* list, CuTexImage* key, int *counter)
{
  int len = list->GetImgWidth();

  key->BindTexture(texDataF4);

  int width = key->GetImgWidth();
  int height = key->GetImgHeight();

  dim3 grid((width + GENERATE_LIST_BLOCK_DIMX - 1)/GENERATE_LIST_BLOCK_DIMX, (height + GENERATE_LIST_BLOCK_DIMY - 1)/GENERATE_LIST_BLOCK_DIMY);
  dim3 block(GENERATE_LIST_BLOCK_DIMX, GENERATE_LIST_BLOCK_DIMY);

  // shared memory is used to store warp scan data
  ListGen_Kernel<<<grid, block, 2*32*sizeof(int)>>>((int4*)list->_cuData, len, width, height, counter);
}

#else

// input list (x, y) (x, y) ....
void ProgramCU::GenerateList(CuTexImage* list, CuTexImage* hist)
{
  int len = list->GetImgWidth();

  list->BindTexture(texDataList);
  hist->BindTexture(texDataI4);

  dim3  grid((len + LISTGEN_BLOCK_DIM -1) / LISTGEN_BLOCK_DIM);
  dim3  block(LISTGEN_BLOCK_DIM);

  ListGen_Kernel<<<grid, block>>>((int4*)list->_cuData, len, hist->GetImgWidth());
}

#endif // GENERATE_FEATURE_LIST_USING_ATOMICS && (GPU_HESSIAN || GPU_SIFT_MODIFIED)

void __global__ ComputeOrientation_Kernel(float4* d_list, int list_len, int width, int height, float sigma, float sigma_step, float gaussian_factor, float sample_factor,
										  int num_orientation, int existing_keypoint, int subpixel, int keepsign
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
                                          , bool doHalfSIFT
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
                                          )
{
  const float ten_degree_per_radius = 5.7295779513082320876798154814105;
  const float radius_per_ten_degrees = 1.0 / 5.7295779513082320876798154814105;
  int idx = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;

  if(idx >= list_len)
    return;

  float4 key;

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  union {
    float        flt; // include response (16b), unused (14b), feature type (2b)
    unsigned int uint;
  } additionalData;

  additionalData.flt = 0.0f;
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  if(existing_keypoint)
  {
    key = tex1Dfetch(texDataF4, idx);
	// read the data
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
 	// unpack scale, x, and y
	// later store just the strongest computed orientation

    // existing keypoint input
    //  key.x: response 8b H (cleared to zero) | x 24b-14.10
    //  key.y: response 8b L (cleared to zero) | y 24b-14.10
    //  key.z: 2b type | 14b unused | scale 16b-8.8 
    //  key.w: orientation (if any)

    // extract x position
    unsigned int tmpValue = *((unsigned int *)(&key.x)); // __float_as_uint(key.x); // CUDA 7.5

    tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
    key.x = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

    // extract y position
    tmpValue = *((unsigned int *)(&key.y)); // __float_as_uint(key.y); // CUDA 7.5

    tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
    key.y = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

    // extract scale
    tmpValue = *((unsigned int *)(&key.z)); // __float_as_uint(key.z); // CUDA 7.5

    tmpValue = tmpValue & FIXED_POINT_SCALE_MASK;
    key.z = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_SCALE_PRECISION_BITS);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
  }
  else
  {
    int4 ikey = tex1Dfetch(texDataList, idx);
    key.x = ikey.x + 0.5f;
    key.y = ikey.y + 0.5f;
    key.z = sigma;

#if !defined GPU_HESSIAN && !defined GPU_SIFT_MODIFIED
    if(subpixel || keepsign)
    {
#endif // !GPU_HESSIAN && !GPU_SIFT_MODIFIED
      // offset: x(response 16b | 14b unused | 2b type), dx, dy, ds
      float4 offset = tex1Dfetch(texDataF4, IMUL(width, ikey.y) + ikey.x);

      if(subpixel)
      {
        key.x += offset.y;
        key.y += offset.z;
        key.z *= pow(sigma_step, offset.w);
      }
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
      additionalData.flt = offset.x;
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

#if !defined GPU_HESSIAN && !defined GPU_SIFT_MODIFIED
      if(keepsign) // not supported for hessian
        key.z *= offset.x;
#endif // !GPU_HESSIAN && !GPU_SIFT_MODIFIED
#if !defined GPU_HESSIAN && !defined GPU_SIFT_MODIFIED
    }
#endif // !GPU_HESSIAN && !GPU_SIFT_MODIFIED
  }

  if(num_orientation == 0)
  {
    key.w = 0;

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    goto key_store_finish;
#else
    d_list[idx] = key;
    return;
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
  }

  float vote[37];
  float gsigma = key.z * gaussian_factor;
  float win = fabs(key.z) * sample_factor;
  float dist_threshold = win * win + 0.5;
  float factor = -0.5f / (gsigma * gsigma);
  float xmin = max(1.5f, floor(key.x - win) + 0.5f);
  float ymin = max(1.5f, floor(key.y - win) + 0.5f);
  float xmax = min(width - 1.5f, floor(key.x + win) + 0.5f);
  float ymax = min(height -1.5f, floor(key.y + win) + 0.5f);

#pragma unroll
  for(int i = 0; i < 36; ++i)
    vote[i] = 0.0f;

  for(float y = ymin; y <= ymax; y += 1.0f)
  {
    float dy = y - key.y;
    dy *= dy;

    for(float x = xmin; x <= xmax; x += 1.0f)
    {
      float dx = x - key.x;
      float sq_dist  = dx * dx + dy;

      if(sq_dist >= dist_threshold)
        continue;

      float2 got = tex2D(texDataF2, x, y);
      // float weight = got.x * exp(sq_dist * factor);
      // float fidx = floorf(got.y * ten_degree_per_radius);
      // int oidx = fidx;
      int oidx = (int)floorf(got.y * ten_degree_per_radius);

      if(oidx < 0)
        oidx += 36;
      vote[oidx] += got.x * expf(sq_dist * factor); // vote[oidx] += weight;
    }
  }

  // filter the vote
  const float one_third = 1.0 / 3.0;

#pragma unroll
  for(int i = 0; i < 6; ++i)
  {
    vote[36] = vote[0];
    float pre = vote[35];

#pragma unroll
    for(int j = 0; j < 36; ++j)
    {
      float temp = one_third * (pre + vote[j] + vote[j + 1]);
      pre = vote[j];
      vote[j] = temp;
    }
  }

  vote[36] = vote[0];

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  if(doHalfSIFT)
  {
#pragma unroll
    for(int i = 0; i < 18; i++)
    {
      vote[i] += vote[i+18];
      vote[i+18] = 0;
    }
  }

  int orientationsCount = 0;
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  // just one orientation
  if((num_orientation == 1) || existing_keypoint)
  {
    int index_max = 0;
    float max_vote = vote[0];

#pragma unroll
    for(int i = 1; i < 36; ++i)
    {
      index_max = (vote[i] > max_vote) ? i : index_max;
      max_vote = max(max_vote, vote[i]);
    }

    float pre = vote[(index_max == 0) ? 35 : index_max - 1];
    float next = vote[index_max + 1];
    float weight = max_vote;
    float off =  0.5f * FDIV(next - pre, weight + weight - next - pre);

    key.w = radius_per_ten_degrees * (index_max + 0.5f + off);

#if !defined GPU_HESSIAN && !defined GPU_SIFT_MODIFIED
    d_list[idx] = key;	
#endif // !GPU_HESSIAN && !GPU_SIFT_MODIFIED
  }
  // multi-orientations allowed
  else
  {
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED

// up to 4 orientations may be stored
// max number of stored orientations depends on num_orientations parameter (1..4)
#define MAX_ORIENTATIONS 4

    // find the maximum value
    float max_vote = vote[0];

#pragma unroll
    for(int i = 1; i < 36; ++i)
      max_vote = max(max_vote, vote[i]);

    float vote_threshold = max_vote * 0.8f;
    float pre = vote[35];

    float max_vot[MAX_ORIENTATIONS+1];
    float max_rot[MAX_ORIENTATIONS+1];

#pragma unroll
    for(int i=0; i < 36; ++i)
    {
      float next = vote[i + 1];

      if((vote[i] > vote_threshold) && (vote[i] > pre) && (vote[i] > next)) // max from neighbours
      {
        float di = 0.5f * FDIV(next - pre, vote[i] + vote[i] - next - pre);
        float rot =  i + di + 0.5f;
        float weight = vote[i];

        int idx = orientationsCount;
        if(orientationsCount > 0)
        {
          // shift values
          while((idx > 0) && (max_vot[idx-1] < weight)) {
            max_vot[idx] = max_vot[idx-1];
            max_rot[idx] = max_rot[idx-1];
            idx--;
          }
        }
        // store maximum found
        max_vot[idx] = weight;
        max_rot[idx] = rot;

        if(orientationsCount < MAX_ORIENTATIONS)
          orientationsCount++;
      }
      pre = vote[i];
    }

    unsigned int packedOrientations = 0;

    // first 4 orientations (if exist)
    unsigned int maxCount = min(4, orientationsCount);
	int idx = 0;
    for(; idx < maxCount; idx++)
    {
      float orientation = max_rot[idx] / 36.0f; 
      if(orientation < 0)
        orientation += 1.0f;

      unsigned int uiOrientation = (unsigned int) floorf(orientation * 255.0f);
      packedOrientations = packedOrientations | (uiOrientation << 8*idx);
    }

    key.w = *((float *)(&packedOrientations)); // __uint_as_float(packedOrientations); // CUDA 7.5

#else
    float max_vote = vote[0];

#pragma unroll
    for(int i = 1; i < 36; ++i)
      max_vote = max(max_vote, vote[i]);

    float vote_threshold = max_vote * 0.8f;
    float pre = vote[35];
    float max_rot[2], max_vot[2] = {0, 0};
    int ocount = 0;

#pragma unroll
    for(int i=0; i < 36; ++i)
    {
      float next = vote[i + 1];

      if((vote[i] > vote_threshold) && (vote[i] > pre) && (vote[i] > next))
      {
        float di = 0.5f * FDIV(next - pre, vote[i] + vote[i] - next - pre);
        float rot = i + di + 0.5f;
        float weight = vote[i];
        ///
        if(weight > max_vot[1])
        {
          if(weight > max_vot[0])
          {
            max_vot[1] = max_vot[0];
            max_rot[1] = max_rot[0];
            max_vot[0] = weight;
            max_rot[0] = rot;
          }
          else
          {
            max_vot[1] = weight;
            max_rot[1] = rot;
          }
          ocount ++;
        }
      }
      pre = vote[i];
    }

    float fr1 = max_rot[0] / 36.0f; 
    if(fr1 < 0)
      fr1 += 1.0f;

    unsigned short us1 = (ocount == 0) ? 65535 : ((unsigned short)floor(fr1 * 65535.0f));
    unsigned short us2 = 65535;

    if(ocount > 1)
    {
      float fr2 = max_rot[1] / 36.0f;

      if(fr2 < 0)
        fr2 += 1.0f;

      us2 = (unsigned short) floor(fr2 * 65535.0f);
    }

    unsigned int uspack = (us2 << 16) | us1;
    key.w = *((float *)(&uspack)); // __uint_as_float(uspack); // CUDA 7.5

#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED


#if !defined GPU_HESSIAN && !defined GPU_SIFT_MODIFIED
    d_list[idx] = key;
#endif // !GPU_HESSIAN && !GPU_SIFT_MODIFIED
  }

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
key_store_finish:

  // input:
  //  additionalData: response 16b | 14b unused | 2b type 

  // output in the feature list d_list
  //  key.x: response 8b H | x 24b-14.10
  //  key.y: response 8b L | y 24b-14.10
  //  key.z: 2b type | 3b orientations count | 11b unused | scale 16b-8.8
  //  key.w: 8b orientation1 | 8b orientation2 | 8b orientation3 | 8b orientation4

  if(!existing_keypoint)
  {
    unsigned int posX = (unsigned int)FLOAT_TO_FIXED_POINT(key.x, FIXED_POINT_POSITION_PRECISION_BITS);
    posX = posX & FIXED_POINT_POSITION_MASK;
    unsigned int posY = (unsigned int)FLOAT_TO_FIXED_POINT(key.y, FIXED_POINT_POSITION_PRECISION_BITS);
    posY = posY & FIXED_POINT_POSITION_MASK;

    // store response
    posX = posX | (additionalData.uint & FIXED_POINT_RESPONSE_MASK);
    posY = posY | ((additionalData.uint << 8) & FIXED_POINT_RESPONSE_MASK);

    unsigned int scale = (unsigned int)(FLOAT_TO_FIXED_POINT(key.z, FIXED_POINT_SCALE_PRECISION_BITS));
    scale = scale & FIXED_POINT_SCALE_MASK;

    // type & orientations count (0 means single float value in key.w, otherwise we have to unpack 8b orientations into floats) 
    scale = scale | ((additionalData.uint & 0x00000003u) << 30) | ((orientationsCount & 0x00000007u) << 27);

    key.z = *((float *)(&scale)); // __uint_as_float(scale); // CUDA 7.5
    key.x = *((float *)(&posX));  // __uint_as_float(posX);  // CUDA 7.5
    key.y = *((float *)(&posY));  // __uint_as_float(posY);  // CUDA 7.5
	
	d_list[idx] = key;
  }
  else
  {
    // for existing keypoint just overwrite computed orientation and other components remain the same (x, y, and scale)
    // ReshapeFeatureListCPU() is not called when we have just one orientation
    d_list[idx].w = key.w;
  }

#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
}

void ProgramCU::ComputeOrientation(CuTexImage* list, CuTexImage* got, CuTexImage* key, float sigma, float sigma_step, int existing_keypoint)
{
  int len = list->GetImgWidth();
  if(len <= 0)
    return;

  int width = got->GetImgWidth();
  int height = got->GetImgHeight();

  if(existing_keypoint)
  {
    list->BindTexture(texDataF4);
  }
  else
  {
    list->BindTexture(texDataList);
#if !defined GPU_HESSIAN && !defined GPU_SIFT_MODIFIED
    if(GlobalUtil::_SubpixelLocalization)
#endif // !GPU_HESSIAN && !GPU_SIFT_MODIFIED
      key->BindTexture(texDataF4);
  }
  got->BindTexture2D(texDataF2);

  const int block_width = (len < ORIENTATION_COMPUTE_PER_BLOCK) ? 16 : ORIENTATION_COMPUTE_PER_BLOCK;
  dim3 grid((len + block_width -1) / block_width);
  dim3 block(block_width);

  ComputeOrientation_Kernel<<<grid, block>>>(
      (float4*) list->_cuData,
      len, width, height, sigma, sigma_step,
      GlobalUtil::_OrientationGaussianFactor,
      GlobalUtil::_OrientationGaussianFactor * GlobalUtil::_OrientationWindowFactor,
      GlobalUtil::_FixedOrientation ? 0 : GlobalUtil::_MaxOrientation,
      existing_keypoint, GlobalUtil::_SubpixelLocalization, GlobalUtil::_KeepExtremumSign
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
      , GlobalUtil::_HalfSIFT
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
      );

  ProgramCU::CheckErrorCUDA("ComputeOrientation");
}

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
template <bool DYNAMIC_INDEXING, bool HALF_SIFT>
#else
template <bool DYNAMIC_INDEXING>
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
void __global__ ComputeDescriptor_Kernel(float4* d_des, int num, int width, int height, float window_factor)
{
  const float rpi = 4.0 / PI;
  int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  int fidx = idx >> 4;

  if(fidx >= num)
    return;

  // fetch the feature
  float4 key = tex1Dfetch(texDataF4, fidx);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  // input in the feature list
  //  key.x: response 8b H | x 24b-14.10
  //  key.y: response 8b L | y 24b-14.10
  //  key.z: 2b type | 14b unused | scale 16b-8.8 
  //  key.w: orientation

  // extract x position
  unsigned int tmpValue = *((unsigned int *)(&key.x)); // __float_as_uint(key.x); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.x = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

  // extract y position
  tmpValue = *((unsigned int *)(&key.y)); // __float_as_uint(key.y); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.y = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

  // extract scale
  tmpValue = *((unsigned int *)(&key.z)); // __float_as_uint(key.z); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_SCALE_MASK;
  key.z = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_SCALE_PRECISION_BITS);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  int bidx = idx & 0xf;
  int ix = bidx & 0x3;
  int iy = bidx >> 2;
  float spt = fabs(key.z * window_factor);
  float s, c;
  
  __sincosf(key.w, &s, &c);

  float anglef = (key.w > PI) ? (key.w - (2.0 * PI)) : key.w;
  float cspt = c * spt;
  float sspt = s * spt;
  float crspt = c / spt;
  float srspt = s / spt;
  float2 offsetpt, pt;
  float xmin, ymin, xmax, ymax, bsz;

  offsetpt.x = ix - 1.5f;
  offsetpt.y = iy - 1.5f;
  pt.x = cspt * offsetpt.x - sspt * offsetpt.y + key.x;
  pt.y = cspt * offsetpt.y + sspt * offsetpt.x + key.y;
  bsz =  fabs(cspt) + fabs(sspt);
  xmin = max(1.5f, floor(pt.x - bsz) + 0.5f);
  ymin = max(1.5f, floor(pt.y - bsz) + 0.5f);
  xmax = min(width - 1.5f, floor(pt.x + bsz) + 0.5f);
  ymax = min(height - 1.5f, floor(pt.y + bsz) + 0.5f);

  float des[9];

#pragma unroll
  for(int i = 0; i < 9; ++i)
    des[i] = 0.0f;

  for(float y = ymin; y <= ymax; y += 1.0f)
  {
    for(float x = xmin; x <= xmax; x += 1.0f)
    {
      float dx = x - pt.x;
      float dy = y - pt.y;
      float nx = crspt * dx + srspt * dy;
      float ny = crspt * dy - srspt * dx;
      float nxn = fabs(nx);
      float nyn = fabs(ny);

      if((nxn < 1.0f) && (nyn < 1.0f))
      {
        float2 cc = tex2D(texDataF2, x, y);

        float dnx = nx + offsetpt.x;
        float dny = ny + offsetpt.y;
        float ww = expf(-0.125f * (dnx * dnx + dny * dny));
        float wx = 1.0 - nxn;
        float wy = 1.0 - nyn;
        float weight = ww * wx * wy * cc.x;
        float theta = (anglef - cc.y) * rpi;

        if(theta < 0)
          theta += 8.0f;

        float fo = floorf(theta);
        int fidx = fo;
        float weight1 = fo + 1.0f  - theta;
        float weight2 = theta - fo;

        if(DYNAMIC_INDEXING)
        {
          des[fidx] += (weight1 * weight);
          des[fidx + 1] += (weight2 * weight);
          // this dynamic indexing part might be slow
        }
        else
        {
#pragma unroll
          for(int k = 0; k < 8; ++k)
          {
            if(k == fidx) 
            {
              des[k] += (weight1 * weight);
              des[k+1] += (weight2 * weight);
            }
          }
        }
      }
    }
  }
  des[0] += des[8];

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  if(HALF_SIFT)
  {
    // half sift -> 4 directions only
    des[0] += des[4];
    des[1] += des[5];
    des[2] += des[6];
    des[3] += des[7];

    d_des[idx] = make_float4(des[0], des[1], des[2], des[3]);

    return;
  }
  else
  {
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  // full sift -> 8 directions
  int didx = idx << 1;

  d_des[didx] = make_float4(des[0], des[1], des[2], des[3]);
  d_des[didx+1] = make_float4(des[4], des[5], des[6], des[7]);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  }
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
}

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
template <bool DYNAMIC_INDEXING, bool HALF_SIFT>
#else
template <bool DYNAMIC_INDEXING>
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
void __global__ ComputeDescriptorRECT_Kernel(float4* d_des, int num, int width, int height, float window_factor)
{
  const float rpi = 4.0 / PI;
  int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  int fidx = idx >> 4;

  if(fidx >= num)
    return;

  // fetch the feature
  float4 key = tex1Dfetch(texDataF4, fidx);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  // input in the feature list
  //  key.x: response 8b H | x 24b-14.10
  //  key.y: response 8b L | y 24b-14.10
  //  key.z: 2b type | 14b unused | scale 16b-8.8 
  //  key.w: orientation1 orientation2

  // extract x position
  unsigned int tmpValue = *((unsigned int *)(&key.x)); // __float_as_uint(key.x); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.x = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

  // extract y position
  tmpValue = *((unsigned int *)(&key.y)); // __float_as_uint(key.y); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.y = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

  // extract scale
  tmpValue = *((unsigned int *)(&key.z)); // __float_as_uint(key.z); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_SCALE_MASK;
  key.z = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_SCALE_PRECISION_BITS);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  int bidx = idx & 0xf;
  int ix = bidx & 0x3;
  int iy = bidx >> 2;
  // float aspect_ratio = key.w / key.z;
  // float aspect_sq = aspect_ratio * aspect_ratio;
  float sptx = key.z * 0.25;
  float spty = key.w * 0.25;
  float xmin, ymin, xmax, ymax;
  float2 pt;

  pt.x = sptx * (ix + 0.5f) + key.x;
  pt.y = spty * (iy + 0.5f) + key.y;

  xmin = max(1.5f, floorf(pt.x - sptx) + 0.5f);
  ymin = max(1.5f, floorf(pt.y - spty) + 0.5f);
  xmax = min(width - 1.5f, floorf(pt.x + sptx) + 0.5f);
  ymax = min(height - 1.5f, floorf(pt.y + spty) + 0.5f);

  float des[9];

#pragma unroll
  for(int i =0; i < 9; ++i)
    des[i] = 0.0f;

  for(float y = ymin; y <= ymax; y += 1.0f)
  {
    for(float x = xmin; x <= xmax; x += 1.0f)
    {
      float nx = (x - pt.x) / sptx;
      float ny = (y - pt.y) / spty;
      float nxn = fabs(nx);
      float nyn = fabs(ny);

      if((nxn < 1.0f) && (nyn < 1.0f))
      {
        float2 cc = tex2D(texDataF2, x, y);

        float wx = 1.0 - nxn;
        float wy = 1.0 - nyn;
        float weight =  wx * wy * cc.x;
        float theta = (- cc.y) * rpi;

        if(theta < 0)
          theta += 8.0f;

        float fo = floorf(theta);
        int fidx = fo;
        float weight1 = fo + 1.0f  - theta;
        float weight2 = theta - fo;

        if(DYNAMIC_INDEXING)
        {
          des[fidx] += (weight1 * weight);
          des[fidx + 1] += (weight2 * weight);
          // this dynamic indexing part might be slow
        }
        else
        {
#pragma unroll
          for(int k = 0; k < 8; ++k)
          {
            if(k == fidx) 
            {
              des[k] += (weight1 * weight);
              des[k+1] += (weight2 * weight);
            }
          }
        }
      }
    }
  }
  des[0] += des[8];

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  if(HALF_SIFT)
  {
    // half sift -> 4 directions only
    des[0] += des[4];
    des[1] += des[5];
    des[2] += des[6];
    des[3] += des[7];

    d_des[idx] = make_float4(des[0], des[1], des[2], des[3]);

    return;
  }
  else
  {
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  // full sift -> 8 directions
  int didx = idx << 1;

  d_des[didx] = make_float4(des[0], des[1], des[2], des[3]);
  d_des[didx+1] = make_float4(des[4], des[5], des[6], des[7]);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  }
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
}

#if (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED) && defined NORMALIZE_DESCRIPTOR_PER_WARP

texture<float2, 1, cudaReadModeElementType> texDataH2;

__device__ float ND_WarpReduction(volatile float *sData, int idx)
{
  int warpIdx = idx & (warpSize-1); // index within warp

  // parallel reduction within warp - just half of the warp is active 
  if(warpIdx < 16)
  {
    sData[idx] += sData[idx + 16];
    sData[idx] += sData[idx +  8];
    sData[idx] += sData[idx +  4];
    sData[idx] += sData[idx +  2];
    sData[idx] += sData[idx +  1];
  }

  return sData[idx-warpIdx]; // first thread in the warp
}

// assumes the block size (128, 1, 1) and grid size (Bx, 1, 1)
void __global__ NormalizeDescriptor_Kernel(float4* d_des, int num)
{
  // size => numThreadsPerBlock * sizeof(float); numThreadsPerBlock must be multiple of 32
  extern __shared__ volatile float reductionCache[];

  // int globalIdx = threadIdx.x + IMUL(blockIdx.x + IMUL(blockIdx.y, gridDim.x), blockDim.x); // 2D grid
  int globalIdx = threadIdx.x + IMUL(blockIdx.x, blockDim.x);  // 1D grid
  int localIdx = threadIdx.x;

  while(globalIdx < 32*num)
  {
    // the vector is first normalized to unit length, thus adjusting for changing image contrast
    float4 temp = tex1Dfetch(texDataF4, globalIdx);
    float norm1 = (temp.x*temp.x + temp.y*temp.y + temp.z*temp.z + temp.w*temp.w);

    reductionCache[localIdx] = norm1;
    // __syncthreads(); // threads in warp are always sync
    norm1 = rsqrt(ND_WarpReduction(reductionCache, localIdx));

    // ... then all feature dimensions are thresholded to a maximum value of 0.2
    temp.x = min(0.2f, temp.x * norm1);
    temp.y = min(0.2f, temp.y * norm1);
    temp.z = min(0.2f, temp.z * norm1);
    temp.w = min(0.2f, temp.w * norm1);

    float norm2 = (temp.x*temp.x + temp.y*temp.y + temp.z*temp.z + temp.w*temp.w);
 
    reductionCache[localIdx] = norm2;
    // __syncthreads(); // threads in warp are always sync
    norm2 = rsqrt(ND_WarpReduction(reductionCache, localIdx));

    // ... and the vector is again normalized to unit length
    temp.x *= norm2;
    temp.y *= norm2;
    temp.z *= norm2;
    temp.w *= norm2;
    d_des[globalIdx] = temp;

    // move to the next descriptor, if there is any unprocessed
    globalIdx += IMUL(gridDim.x, blockDim.x);
  }
}

// assumes the block size (128, 1, 1) and grid size (Bx, 1, 1)
// version used for half sift
void __global__ NormalizeDescriptor_Kernel(float2* d_des, int num)
{
  // size => numThreadsPerBlock * sizeof(float); numThreadsPerBlock must be multiple of 32
  extern __shared__ volatile float reductionCache[];

  // int globalIdx = threadIdx.x + IMUL(blockIdx.x + IMUL(blockIdx.y, gridDim.x), blockDim.x); // 2D grid
  int globalIdx = threadIdx.x + IMUL(blockIdx.x, blockDim.x);  // 1D grid
  int localIdx = threadIdx.x;

  while(globalIdx < 32*num)
  {
    // the vector is first normalized to unit length, thus adjusting for changing image contrast
    float2 temp = tex1Dfetch(texDataH2, globalIdx);
    float norm1 = (temp.x*temp.x + temp.y*temp.y);

    reductionCache[localIdx] = norm1;
    // __syncthreads(); // threads in warp are always sync
    norm1 = rsqrt(ND_WarpReduction(reductionCache, localIdx));

    // ... then all feature dimensions are thresholded to a maximum value of 0.2
    temp.x = min(0.2f, temp.x * norm1);
    temp.y = min(0.2f, temp.y * norm1);

    float norm2 = (temp.x*temp.x + temp.y*temp.y);
 
    reductionCache[localIdx] = norm2;
    // __syncthreads(); // threads in warp are always sync
    norm2 = rsqrt(ND_WarpReduction(reductionCache, localIdx));

    // ... and the vector is again normalized to unit length
    temp.x *= norm2;
    temp.y *= norm2;
    d_des[globalIdx] = temp;

    // move to the next descriptor, if there is any unprocessed
    globalIdx += IMUL(gridDim.x, blockDim.x);
  }
}

#else
template<int DESCRIPTOR_SIZE, int SHIFT>
void __global__ NormalizeDescriptor_Kernel(float4* d_des, int num)
{
  float4 temp[DESCRIPTOR_SIZE];
  int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;

  if(idx >= num)
    return;

  int sidx = idx << SHIFT;
  float norm1 = 0;
  float norm2 = 0;

#pragma unroll
  // the vector is first normalized to unit length, thus adjusting for changing image contrast
  for(int i = 0; i < DESCRIPTOR_SIZE; ++i)
  {
    temp[i] = tex1Dfetch(texDataF4, sidx+i);
    norm1 += (temp[i].x*temp[i].x + temp[i].y*temp[i].y + temp[i].z*temp[i].z + temp[i].w*temp[i].w);
  }
  norm1 = rsqrt(norm1);

#pragma unroll
  // ... then all feature dimensions are thresholded to a maximum value of 0.2
  for(int i = 0; i < DESCRIPTOR_SIZE; ++i)
  {
    temp[i].x = min(0.2f, temp[i].x * norm1);
    temp[i].y = min(0.2f, temp[i].y * norm1);
    temp[i].z = min(0.2f, temp[i].z * norm1);
    temp[i].w = min(0.2f, temp[i].w * norm1);

    norm2 += (temp[i].x*temp[i].x + temp[i].y*temp[i].y + temp[i].z*temp[i].z + temp[i].w*temp[i].w);
  }
  norm2 = rsqrt(norm2);

#pragma unroll
  // ... and the vector is again normalized to unit length
  for(int i = 0; i < DESCRIPTOR_SIZE; ++i)
  {
    temp[i].x *= norm2;
    temp[i].y *= norm2;
    temp[i].z *= norm2;
    temp[i].w *= norm2;
    d_des[sidx + i] = temp[i];
  }
}
#endif // (GPU_HESSIAN || GPU_SIFT_MODIFIED) && NORMALIZE_DESCRIPTOR_PER_WARP

void ProgramCU::ComputeDescriptor(CuTexImage*list, CuTexImage* got, CuTexImage* dtex, int rect, int stream)
{
  int num = list->GetImgWidth();
  int width = got->GetImgWidth();
  int height = got->GetImgHeight();

  dtex->InitTexture(num * 128, 1, 1);
  got->BindTexture2D(texDataF2);
  list->BindTexture(texDataF4);

  int block_width = DESCRIPTOR_COMPUTE_BLOCK_SIZE;
  dim3 grid((num * 16 + block_width -1) / block_width);
  dim3 block(block_width);

  if(rect)
  {
    if(GlobalUtil::_UseDynamicIndexing)
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    {
      if(GlobalUtil::_HalfSIFT)
        ComputeDescriptorRECT_Kernel<true, true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
      else
        ComputeDescriptorRECT_Kernel<true, false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
    }
#else
      ComputeDescriptorRECT_Kernel<true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
    else
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    {
      if(GlobalUtil::_HalfSIFT)
        ComputeDescriptorRECT_Kernel<false, true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
      else
        ComputeDescriptorRECT_Kernel<false, false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
    }
#else
      ComputeDescriptorRECT_Kernel<false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
  }
  else
  {
    if(GlobalUtil::_UseDynamicIndexing)
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    {
      if(GlobalUtil::_HalfSIFT)
        ComputeDescriptor_Kernel<true, true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
      else
        ComputeDescriptor_Kernel<true, false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
    }
#else
      ComputeDescriptor_Kernel<true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
    else
#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    {
      if(GlobalUtil::_HalfSIFT)
        ComputeDescriptor_Kernel<false, true><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
      else
        ComputeDescriptor_Kernel<false, false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
    }
#else
      ComputeDescriptor_Kernel<false><<<grid, block>>>((float4*) dtex->_cuData, num, width, height, GlobalUtil::_DescriptorWindowFactor);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
  }

  if(GlobalUtil::_NormalizedSIFT)
  {
#if (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED) && defined NORMALIZE_DESCRIPTOR_PER_WARP
	// 32 threads (one warp) normalize one descriptor -> one thread handles one float4
    // 1D block size (128, 1, 1) => four descriptors are normalized per block
    // 1D grid size (Bx, 1, 1)

    if(GlobalUtil::_HalfSIFT)
      dtex->BindTexture(texDataH2);
    else
      dtex->BindTexture(texDataF4);

    const int blockWidth = DESCRIPTOR_NORMALIZE_PER_BLOCK;
    int blocksInGrid = min(16384, (num*32 + blockWidth -1) / blockWidth);
    dim3 grid(blocksInGrid);
    dim3 block(blockWidth);

    if(GlobalUtil::_HalfSIFT)
      NormalizeDescriptor_Kernel<<<grid, block, blockWidth*sizeof(float)>>>((float2*) dtex->_cuData, num);
    else
      NormalizeDescriptor_Kernel<<<grid, block, blockWidth*sizeof(float)>>>((float4*) dtex->_cuData, num);
#else
    dtex->BindTexture(texDataF4);

    const int block_width = DESCRIPTOR_NORMALIZE_PER_BLOCK;
    dim3 grid((num + block_width -1) / block_width);
    dim3 block(block_width);

    NormalizeDescriptor_Kernel<32, 5><<<grid, block>>>((float4*) dtex->_cuData, num);
#endif // (GPU_HESSIAN || GPU_SIFT_MODIFIED) && NORMALIZE_DESCRIPTOR_PER_WARP
  }

  CheckErrorCUDA("ComputeDescriptor");
}

#if (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED) && defined TOP_K_SELECTION

// Enables maximum occupancy - bitonic sort
#define SHARED_SIZE_LIMIT 1024U // CC2.0
#define TOPK_BLOCK_SIZE    128  // 256

void ProgramCU::TopKInit(TopKData &data, int listSize, int countThreshold)
{
  data.keypointsCount = listSize;

  // the closest power of two higher than keypoints count
  data.keypointsCountAsPowerOfTwo = 1;
  while (data.keypointsCountAsPowerOfTwo < listSize)
    data.keypointsCountAsPowerOfTwo *= 2;

  // keypoints count have to be at least SHARED_SIZE_LIMIT (required by bitonic sort implementation)
  data.keypointsCountAsPowerOfTwo = max(data.keypointsCountAsPowerOfTwo, SHARED_SIZE_LIMIT);

  // allocate arrays to store responses and indices
  cudaMalloc(&data.keys, (data.keypointsCountAsPowerOfTwo+1) * sizeof(float));           // one extra element in both arrays is used by prefix scan
  cudaMalloc(&data.indices, (data.keypointsCountAsPowerOfTwo+1) * sizeof(unsigned int));

  data.topKCountThreshold = countThreshold;
}

void ProgramCU::TopKFinish(TopKData &data)
{
  // release all data
  cudaFree(data.keys);
  cudaFree(data.indices);

  cudaFree(data.devLevelFeaturesCount);
  delete[] data.levelFeaturesCount;
}

void __global__ CopyTopKData_Kernel(int width, float *keys, unsigned int *indices, int listLen, int offset, int keypointsCount)
{
  int idx = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;

  if(idx >= listLen)
    return;

  float response = MIN_VALUE;

  if(idx+offset < keypointsCount)
  {
    // read keypoint index
    int4 ikey = tex1Dfetch(texDataList, idx);

    int keyIdx = IMUL(width, ikey.y) + ikey.x;
    // read keypoint additional info
    float4 key = tex1Dfetch(texDataF4, keyIdx);

 #if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    // keypoint info
    //  key.x: response 16b | 14b unused | 2b type
    //  key.y: x
    //  key.z: y 
    //  key.w: scale

    // extract response
    unsigned int value = *((unsigned int *)(&key.x)); // __float_as_uint(key.x); // CUDA 7.5

    value = (value & 0xFFFF0000u) >> 16;
    response = __half2float((unsigned short)value);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
  }

  keys[idx+offset] = fabs(response);
  indices[idx+offset] = idx+offset;
}

void ProgramCU::TopKCopyData(CuTexImage* list, CuTexImage* key, TopKData &topKData, int offset)
{
  bool isActivePadding = ((list == NULL) && (key == NULL));    // clear the padding values?
  int len = isActivePadding ? (topKData.keypointsCountAsPowerOfTwo - offset) : list->GetImgWidth();

  if(len <= 0)
    return;

  int width = isActivePadding ? 0 : key->GetImgWidth();

  if(!isActivePadding)
  {
    list->BindTexture(texDataList);

    key->BindTexture(texDataF4);
  }

  int blockWidth = TOPK_BLOCK_SIZE;
  dim3 grid((len + blockWidth - 1) / blockWidth);
  dim3 block(blockWidth);

  CopyTopKData_Kernel<<<grid, block>>>(width, topKData.keys, topKData.indices, len, offset, topKData.keypointsCount);

  CheckErrorCUDA("CopyTopKData");
}

// Map to single instructions on G8x / G9x / G100
#define UMUL(a, b) __umul24((a), (b))
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

__device__ inline void BMS_Comparator(float &keyA, unsigned int &valA, float &keyB, unsigned int &valB, unsigned int dir)
{
  union {
    unsigned int uintValue;
	float floatValue;
  };

  if ((keyA > keyB) == dir)
  {
    floatValue = keyA; keyA = keyB; keyB = floatValue;
    uintValue = valA; valA = valB; valB = uintValue;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Monolithic bitonic sort kernel for short arrays fitting into shared memory
////////////////////////////////////////////////////////////////////////////////
__global__ void BitonicSortShared_Kernel(float *dstKey, unsigned int *dstVal, float *srcKey, unsigned int *srcVal, unsigned int arrayLength, unsigned int dir)
{
  // Shared memory storage for one or more short vectors
  __shared__ float sKey[SHARED_SIZE_LIMIT];
  __shared__ unsigned int sVal[SHARED_SIZE_LIMIT];

  // Offset to the beginning of subbatch and load data
  int offset = blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x; 
  srcKey += offset;
  srcVal += offset;
  dstKey += offset;
  dstVal += offset;

  sKey[threadIdx.x +                       0] = srcKey[                      0];
  sVal[threadIdx.x +                       0] = srcVal[                      0];
  sKey[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = srcKey[(SHARED_SIZE_LIMIT / 2)];
  sVal[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = srcVal[(SHARED_SIZE_LIMIT / 2)];

  for (unsigned int size = 2; size < arrayLength; size <<= 1)
  {
    // bitonic merge
    unsigned int ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);

    for (unsigned int stride = size / 2; stride > 0; stride >>= 1)
    {
      __syncthreads();

      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      BMS_Comparator(
        sKey[pos +      0], sVal[pos +      0],
        sKey[pos + stride], sVal[pos + stride],
        ddd
      );
    }
  }

  // ddd == dir for the last bitonic merge step
  {
    for (unsigned int stride = arrayLength / 2; stride > 0; stride >>= 1)
    {
      __syncthreads();

      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      BMS_Comparator(
        sKey[pos +      0], sVal[pos +      0],
        sKey[pos + stride], sVal[pos + stride],
        dir
      );
    }
  }

  __syncthreads();

  dstKey[                      0] = sKey[threadIdx.x +                       0];
  dstVal[                      0] = sVal[threadIdx.x +                       0];
  dstKey[(SHARED_SIZE_LIMIT / 2)] = sKey[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  dstVal[(SHARED_SIZE_LIMIT / 2)] = sVal[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

////////////////////////////////////////////////////////////////////////////////
// Bitonic sort kernel for large arrays (not fitting into shared memory)
////////////////////////////////////////////////////////////////////////////////
// Bottom-level bitonic sort
// Almost the same as bitonicSortShared with the exception of even / odd subarrays being sorted in opposite directions
// Bitonic merge accepts both Ascending | descending or descending | ascending sorted pairs
__global__ void BitonicSortShared1_Kernel(float *dstKey, unsigned int *dstVal, float *srcKey, unsigned int *srcVal)
{
  // Shared memory storage for current subarray
  __shared__ float sKey[SHARED_SIZE_LIMIT];
  __shared__ unsigned int sVal[SHARED_SIZE_LIMIT];

  // Offset to the beginning of subarray and load data
  int offset = blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x; 
  srcKey += offset;
  srcVal += offset;
  dstKey += offset;
  dstVal += offset;

  sKey[threadIdx.x +                       0] = srcKey[                      0];
  sVal[threadIdx.x +                       0] = srcVal[                      0];
  sKey[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = srcKey[(SHARED_SIZE_LIMIT / 2)];
  sVal[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = srcVal[(SHARED_SIZE_LIMIT / 2)];

  for (unsigned int size = 2; size < SHARED_SIZE_LIMIT; size <<= 1)
  {
    // Bitonic merge
    unsigned int ddd = (threadIdx.x & (size / 2)) != 0;

    for (unsigned int stride = size / 2; stride > 0; stride >>= 1)
    {
      __syncthreads();

      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      BMS_Comparator(
        sKey[pos +      0], sVal[pos +      0],
        sKey[pos + stride], sVal[pos + stride],
        ddd
      );
    }
  }

  // odd / even arrays of SHARED_SIZE_LIMIT elements
  // sorted in opposite directions
  unsigned int ddd = blockIdx.x & 1;
  {
    for (unsigned int stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
    {
      __syncthreads();

      unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      BMS_Comparator(
        sKey[pos +      0], sVal[pos +      0],
        sKey[pos + stride], sVal[pos + stride],
        ddd
      );
    }
  }

  __syncthreads();

  dstKey[                      0] = sKey[threadIdx.x +                       0];
  dstVal[                      0] = sVal[threadIdx.x +                       0];
  dstKey[(SHARED_SIZE_LIMIT / 2)] = sKey[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  dstVal[(SHARED_SIZE_LIMIT / 2)] = sVal[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

// Bitonic merge iteration for stride >= SHARED_SIZE_LIMIT
__global__ void BitonicMergeGlobal_Kernel(float *dstKey, unsigned int *dstVal, float *srcKey, unsigned int *srcVal, unsigned int arrayLength, unsigned int size, unsigned int stride, unsigned int dir)
{
  unsigned int global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int comparatorI = global_comparatorI & (arrayLength / 2 - 1);

  // Bitonic merge
  unsigned int ddd = dir ^ ((comparatorI & (size / 2)) != 0);
  unsigned int pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

  float keyA = srcKey[pos + 0];
  unsigned int valA =  srcVal[pos + 0];
  float keyB = srcKey[pos + stride];
  unsigned int valB =  srcVal[pos + stride];

  BMS_Comparator(
    keyA, valA,
    keyB, valB,
    ddd
  );

  dstKey[pos +      0] = keyA;
  dstVal[pos +      0] = valA;
  dstKey[pos + stride] = keyB;
  dstVal[pos + stride] = valB;
}

// Combined bitonic merge steps for size > SHARED_SIZE_LIMIT and stride = [1 .. SHARED_SIZE_LIMIT / 2]
__global__ void BitonicMergeShared_Kernel(float *dstKey, unsigned int *dstVal, float *srcKey, unsigned int *srcVal, unsigned int arrayLength, unsigned int size, unsigned int dir)
{
  // Shared memory storage for current subarray
  __shared__ float sKey[SHARED_SIZE_LIMIT];
  __shared__ unsigned int sVal[SHARED_SIZE_LIMIT];

  int offset = blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x; 
  srcKey += offset;
  srcVal += offset;
  dstKey += offset;
  dstVal += offset;

  sKey[threadIdx.x +                       0] = srcKey[                      0];
  sVal[threadIdx.x +                       0] = srcVal[                      0];
  sKey[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = srcKey[(SHARED_SIZE_LIMIT / 2)];
  sVal[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] = srcVal[(SHARED_SIZE_LIMIT / 2)];

  // Bitonic merge
  unsigned int comparatorI = UMAD(blockIdx.x, blockDim.x, threadIdx.x) & ((arrayLength / 2) - 1);
  unsigned int ddd = dir ^ ((comparatorI & (size / 2)) != 0);

  for(unsigned int stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1)
  {
    __syncthreads();

    unsigned int pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
    BMS_Comparator(
      sKey[pos +      0], sVal[pos +      0],
      sKey[pos + stride], sVal[pos + stride],
      ddd
    );
  }

  __syncthreads();

  dstKey[                      0] = sKey[threadIdx.x +                       0];
  dstVal[                      0] = sVal[threadIdx.x +                       0];
  dstKey[(SHARED_SIZE_LIMIT / 2)] = sKey[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  dstVal[(SHARED_SIZE_LIMIT / 2)] = sVal[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

unsigned int BMS_FactorOf2(unsigned int *log2L, unsigned int L)
{
  if (!L)
  {
    *log2L = 0;
    return 0;
  }
  else
  {
    for (*log2L = 0; (L & 1) == 0; L >>= 1, *log2L++);

    return L;
  }
}

///////////////////////////////////////////////////////////////////////////////
// bitonic sort is borrowed from the sorting networks sample which is a part of 
// the NVIDIA GPU Computing SDK (NVIDIA CUDA Code Samples)
unsigned int BMS_BitonicSort(float *dstKey, unsigned int *dstVal, float *srcKey, unsigned int *srcVal, unsigned int arrayLength, unsigned int dir)
{
  // Nothing to sort
  if(arrayLength < 2)
    return 0;

  // only power-of-two array lengths are supported by this implementation
  unsigned int log2L;
  unsigned int factorizationRemainder = BMS_FactorOf2(&log2L, arrayLength);
  assert(factorizationRemainder == 1);

  dir = (dir != 0);

  unsigned int blockCount = arrayLength / SHARED_SIZE_LIMIT;
  unsigned int threadCount = SHARED_SIZE_LIMIT / 2;

  if(arrayLength <= SHARED_SIZE_LIMIT)
  {
    assert(arrayLength % SHARED_SIZE_LIMIT == 0);
    BitonicSortShared_Kernel<<<blockCount, threadCount>>>(dstKey, dstVal, srcKey, srcVal, arrayLength, dir);
  }
  else
  {
    BitonicSortShared1_Kernel<<<blockCount, threadCount>>>(dstKey, dstVal, srcKey, srcVal);

    for(unsigned int size = 2 * SHARED_SIZE_LIMIT; size <= arrayLength; size <<= 1)
      for (unsigned stride = size / 2; stride > 0; stride >>= 1)
        if(stride >= SHARED_SIZE_LIMIT)
        {
          BitonicMergeGlobal_Kernel<<<arrayLength / 512, 256>>>(dstKey, dstVal, dstKey, dstVal, arrayLength, size, stride, dir);
        }
        else
        {
          BitonicMergeShared_Kernel<<<blockCount, threadCount>>>(dstKey, dstVal, dstKey, dstVal, arrayLength, size, dir);
          break;
        }
  }

  return threadCount;
}

#undef SHARED_SIZE_LIMIT

void ProgramCU::TopKSort(TopKData &topKData)
{
  // sort in descending order
  unsigned int threadsCount = BMS_BitonicSort(
    topKData.keys, topKData.indices,      // dst 
    topKData.keys, topKData.indices,      // src
    topKData.keypointsCountAsPowerOfTwo,  // array length - have to be power of 2
    0                                     // sort direction
  );
}

__global__ void MarkSelectedElements_Kernel(unsigned int *outFlags, unsigned int *inIndices, int len, int topK)
{
  int threadID = threadIdx.x + blockDim.x* blockIdx.x;

  if (threadID >= len)
    return;

  if(threadID < topK)
  {
    unsigned int index = inIndices[threadID];
    outFlags[index] = 1;
  } // 0 are not written because of cleared array to 0
}

/////////////////////
// Define this to more rigorously avoid bank conflicts, even at the lower (root) levels of the tree
// Note that due to the higher addressing overhead, performance is lower with ZERO_BANK_CONFLICTS enabled.
// It is provided as an example.
//#define ZERO_BANK_CONFLICTS 

// 16 banks on G80, 32 foc CC 2.X and 3.X
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

///////////////////////////////////////////////////////////////////////////////
// prefix scan is borrowed from the scan large array sample which is a part of 
// the NVIDIA GPU Computing SDK  
//
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// https://www.cs.unc.edu/~prins/Classes/633/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// excellent paper "Prefix sums and their applications".
// http://www.cs.cmu.edu/~blelloch/papers/Ble93.pdf
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//

template <bool isNP2>
__device__ void PPS_LoadSharedChunkFromMem(unsigned int *sData, const unsigned int *inData, int n, int baseIndex, int& ai, int& bi, int& memAi, int& memBi, int& bankOffsetA, int& bankOffsetB)
{
  int threadID = threadIdx.x;
  memAi = baseIndex + threadIdx.x;
  memBi = memAi + blockDim.x;

  ai = threadID;
  bi = threadID + blockDim.x;

  // compute spacing to avoid bank conflicts
  bankOffsetA = CONFLICT_FREE_OFFSET(ai);
  bankOffsetB = CONFLICT_FREE_OFFSET(bi);

  // cache the computational window in shared memory, pad values beyond n with zeros
  sData[ai + bankOffsetA] = inData[memAi]; 
    
  if(isNP2) // compile-time decision
  {
    sData[bi + bankOffsetB] = (bi < n) ? inData[memBi] : 0; 
  }
  else
  {
    sData[bi + bankOffsetB] = inData[memBi]; 
  }
}

template <bool isNP2>
__device__ void PPS_StoreSharedChunkToMem(unsigned int *outData, const unsigned int *sData, int n, int ai, int bi, int memAi, int memBi, int bankOffsetA, int bankOffsetB)
{
  __syncthreads();

  // write results to global memory
  outData[memAi] = sData[ai + bankOffsetA]; 
  if(isNP2) // compile-time decision
  {
    if(bi < n)
      outData[memBi] = sData[bi + bankOffsetB]; 
  }
  else
  {
    outData[memBi] = sData[bi + bankOffsetB]; 
  }
}

template <bool storeSum>
__device__ void PPS_ClearLastElement(unsigned int *sData, unsigned int *blockSums, int blockIndex)
{
  if(threadIdx.x == 0)
  {
    int index = (blockDim.x << 1) - 1;
    index += CONFLICT_FREE_OFFSET(index);
        
    if(storeSum) // compile-time decision
    {
      // write this block's total sum to the corresponding index in the blockSums array
      blockSums[blockIndex] = sData[index];
    }

    // zero the last element in the scan so it will propagate back to the front
    sData[index] = 0;
  }
}

__device__ unsigned int PPS_BuildSum(unsigned int *sData)
{
  unsigned int threadID = threadIdx.x;
  unsigned int stride = 1;
    
  // build the sum in place up the tree
  for(int d = blockDim.x; d > 0; d >>= 1)
  {
    __syncthreads();

    if(threadID < d)      
    {
      int i  = IMUL(IMUL(2, stride), threadID);
      int ai = i + stride - 1;
      int bi = ai + stride;

      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      sData[bi] += sData[ai];
    }

    stride *= 2;
  }

  return stride;
}

__device__ void PPS_ScanRootToLeaves(unsigned int *sData, unsigned int stride)
{
  unsigned int threadID = threadIdx.x;

  // traverse down the tree building the scan in place
  for(int d = 1; d <= blockDim.x; d *= 2)
  {
    stride >>= 1;

    __syncthreads();

    if(threadID < d)
    {
      int i = IMUL(IMUL(2, stride), threadID);
      int ai = i + stride - 1;
      int bi = ai + stride;

      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      unsigned int t = sData[ai];
      sData[ai] = sData[bi];
      sData[bi] += t;
    }
  }
}

template <bool storeSum>
__device__ void PPS_PrescanBlock(unsigned int *data, int blockIndex, unsigned int *blockSums)
{
  // build the sum in place up the tree
  int stride = PPS_BuildSum(data);

  PPS_ClearLastElement<storeSum>(data, blockSums, (blockIndex == 0) ? blockIdx.x : blockIndex);
  // traverse down tree to build the scan 
  PPS_ScanRootToLeaves(data, stride);
}

template <bool storeSum, bool isNP2>
__global__ void PPS_Prescan_Kernel(unsigned int *outData, const unsigned int *inData, unsigned int *blockSums, int n, int blockIndex, int baseIndex)
{
  extern __shared__ unsigned int shData[];
  int ai, bi, memAi, memBi, bankOffsetA, bankOffsetB;

  // load data into shared memory
  PPS_LoadSharedChunkFromMem<isNP2>(shData, inData, n, (baseIndex == 0) ? IMUL(blockIdx.x, (blockDim.x << 1)) : baseIndex, ai, bi, memAi, memBi, bankOffsetA, bankOffsetB);
  // scan the data in each block
  PPS_PrescanBlock<storeSum>(shData, blockIndex, blockSums);
  // write results to device memory
  PPS_StoreSharedChunkToMem<isNP2>(outData, shData, n, ai, bi, memAi, memBi, bankOffsetA, bankOffsetB);
}

__global__ void PPS_UniformAdd_Kernel(unsigned int *data, unsigned int *uniforms, int n, int blockOffset, int baseIndex)
{
  __shared__ unsigned int uni;

  if(threadIdx.x == 0)
    uni = uniforms[blockIdx.x + blockOffset];
    
  unsigned int address = IMUL(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x;

  __syncthreads();
    
  // note two adds per thread
  data[address] += uni;
  data[address + blockDim.x] += (threadIdx.x + blockDim.x < n) * uni;
}

inline bool PPS_IsPowerOfTwo(int n)
{
  return ((n&(n-1))==0) ;
}

inline int PPS_FloorPow2(int n)
{
#ifdef WIN32
  // method 2
  return 1 << (int)logb((float)n);
#else
  // method 1
  // float nf = (float)n;
  // return 1 << (((*(int*)&nf) >> 23) - 127); 
  int exp;
  frexp((float)n, &exp);
  return 1 << (exp - 1);
#endif
}

#define THREADS_PER_BLOCK 128

void PPS_PreallocBlockSums(TopKData &topKData, unsigned int maxNumElements)
{
  assert(topKData.numElementsAllocated == 0); // shouldn't be called 

  topKData.numElementsAllocated = maxNumElements;

  unsigned int blockSize = THREADS_PER_BLOCK; // max size of the thread blocks
  unsigned int numElements = maxNumElements;

  int level = 0;

  do
  {       
    unsigned int numBlocks = max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    if(numBlocks > 1)
      level++;

    numElements = numBlocks;
  } while(numElements > 1);

  topKData.scanBlockSums = (unsigned int**) malloc(level * sizeof(unsigned int*));
  topKData.numLevelsAllocated = level;
    
  numElements = maxNumElements;
  level = 0;
    
  do
  {       
    unsigned int numBlocks = max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    if (numBlocks > 1) 
      cudaMalloc((void**) &topKData.scanBlockSums[level++], numBlocks * sizeof(unsigned int));

    numElements = numBlocks;
  } while(numElements > 1);
}

void PPS_DeallocBlockSums(TopKData &topKData)
{
  for(unsigned int i = 0; i < topKData.numLevelsAllocated; i++)
    cudaFree(topKData.scanBlockSums[i]);
    
  free((void**)topKData.scanBlockSums);

  topKData.scanBlockSums = NULL;
  topKData.numElementsAllocated = 0;
  topKData.numLevelsAllocated = 0;
}

// prefix scan is borrowed from the scan large array sample which is a part of NVIDIA GPU Computing SDK  
void PPS_PrescanArrayRecursive(TopKData &topKData, unsigned int *outArray, const unsigned int *inArray, int numElements, int level)
{
  unsigned int blockSize = THREADS_PER_BLOCK; // max size of the thread blocks
  unsigned int numBlocks = max(1, (int)ceil((float)numElements / (2.f * blockSize)));
  unsigned int numThreads;

  if(numBlocks > 1)
    numThreads = blockSize;
  else if(PPS_IsPowerOfTwo(numElements))
    numThreads = numElements / 2;
  else
    numThreads = PPS_FloorPow2(numElements);

  unsigned int numElementsPerBlock = numThreads * 2;

  // if this is a non-power-of-2 array, the last block will be non-full
  // compute the smallest power of 2 able to compute its scan.
  unsigned int numElementsLastBlock = numElements - (numBlocks-1) * numElementsPerBlock;
  unsigned int numThreadsLastBlock = max(1, numElementsLastBlock / 2);
  unsigned int np2LastBlock = 0;
  unsigned int sharedMemLastBlock = 0;
    
  if (numElementsLastBlock != numElementsPerBlock)
  {
    np2LastBlock = 1;

    if(!PPS_IsPowerOfTwo(numElementsLastBlock))
      numThreadsLastBlock = PPS_FloorPow2(numElementsLastBlock);    
        
    unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
    sharedMemLastBlock = sizeof(unsigned int) * (2 * numThreadsLastBlock + extraSpace);
  }

  // padding space is used to avoid shared memory bank conflicts
  unsigned int extraSpace = numElementsPerBlock / NUM_BANKS;
  unsigned int sharedMemSize = sizeof(unsigned int) * (numElementsPerBlock + extraSpace);

  // setup execution parameters
  // if NP2, we process the last block separately
  dim3 grid(max(1, numBlocks - np2LastBlock), 1, 1); 
  dim3 threads(numThreads, 1, 1);

  // execute the scan
  if(numBlocks > 1)
  {
    PPS_Prescan_Kernel<true, false><<<grid, threads, sharedMemSize>>>(outArray, inArray, topKData.scanBlockSums[level], numThreads*2, 0, 0);
    if(np2LastBlock)
      PPS_Prescan_Kernel<true, true><<<1, numThreadsLastBlock, sharedMemLastBlock>>>(outArray, inArray, topKData.scanBlockSums[level], numElementsLastBlock, numBlocks - 1, numElements - numElementsLastBlock);

    // After scanning all the sub-blocks, we are mostly done.  But now we need to take all of the last
	// values of the sub-blocks and scan those. This will give us a new value that must be sdded to each
	// block to get the final results.
    // recursive (CPU) call
    PPS_PrescanArrayRecursive(topKData, topKData.scanBlockSums[level], topKData.scanBlockSums[level], numBlocks, level+1);

    PPS_UniformAdd_Kernel<<<grid, threads>>>(outArray, topKData.scanBlockSums[level], numElements - numElementsLastBlock, 0, 0);
    if(np2LastBlock)
      PPS_UniformAdd_Kernel<<<1, numThreadsLastBlock>>>(outArray, topKData.scanBlockSums[level], numElementsLastBlock, numBlocks - 1, numElements - numElementsLastBlock);
  }
  else if(PPS_IsPowerOfTwo(numElements))
  {
    PPS_Prescan_Kernel<false, false><<<grid, threads, sharedMemSize>>>(outArray, inArray, 0, numThreads*2, 0, 0);
  }
  else
  {
    PPS_Prescan_Kernel<false, true><<<grid, threads, sharedMemSize>>>(outArray, inArray, 0, numElements, 0, 0);
  }
}

void ProgramCU::TopKPrefixScan(TopKData &topKData)
{
  if(topKData.keypointsCount <= 0)
    return;

  topKData.numElementsAllocated = 0;
  topKData.numLevelsAllocated = 0;

  unsigned int numElements = topKData.keypointsCountAsPowerOfTwo;

  unsigned int *devIdxs = (unsigned int *)topKData.keys;
  unsigned int *devData = topKData.indices;

  // clear keypoints flags to 0 
  cudaMemset(devIdxs, 0, (numElements+1)*sizeof(unsigned int));

  const int blocks = (numElements + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

  // prepare input for prefix scan - mark selected keypoints by 1
  MarkSelectedElements_Kernel<<<blocks, THREADS_PER_BLOCK>>>(devIdxs, devData, numElements+1, topKData.topKCountThreshold);

  PPS_PreallocBlockSums(topKData, numElements);

  // run the prescan
  PPS_PrescanArrayRecursive(topKData, devIdxs, devIdxs, numElements, 0);

  PPS_DeallocBlockSums(topKData);    

  CheckErrorCUDA("TopKPrefixScan");
}

#undef THREADS_PER_BLOCK

void __global__ GetLevelsFeatureNum_Kernel(unsigned int *indices, int *levelFeatureNum, int levelsCount)
{
  int threadID = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;

  if(threadID >= levelsCount)
    return;

  unsigned int index = levelFeatureNum[threadID];

  levelFeatureNum[threadID] = indices[index];
}

void ProgramCU::TopKGetLevelsFeatureNum(TopKData &topKData)
{
  if(topKData.levelsCount <= 0)
    return;

  cudaMalloc((void**)&(topKData.devLevelFeaturesCount), topKData.levelsCount*sizeof(unsigned int));
  cudaMemcpy(topKData.devLevelFeaturesCount, topKData.levelFeaturesCount, topKData.levelsCount*sizeof(unsigned int), cudaMemcpyHostToDevice);

  int blockWidth = TOPK_BLOCK_SIZE;
  dim3 grid((topKData.levelsCount + blockWidth - 1) / blockWidth);
  dim3 block(blockWidth);

  GetLevelsFeatureNum_Kernel<<<grid, block>>>((unsigned int *)topKData.keys, topKData.devLevelFeaturesCount, topKData.levelsCount);

  cudaMemcpy(topKData.levelFeaturesCount, topKData.devLevelFeaturesCount, topKData.levelsCount*sizeof(unsigned int), cudaMemcpyDeviceToHost);

  CheckErrorCUDA("TopKGetLevelsFeatureNum");
}

void __global__ CompactLevelFeatures_Kernel(float4 *outFeatures, unsigned int *indices, unsigned int offset, unsigned int featuresCount)
{
  int threadID = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;

  if(threadID >= featuresCount)
    return;

  unsigned int index = indices[offset + threadID];
  if(index == indices[offset + threadID + 1])
    return;

  unsigned int firstIndex = indices[offset];
  index -= firstIndex;

  float4 key = tex1Dfetch(texDataF4, threadID);

  outFeatures[index] = key;
}

void ProgramCU::TopKCompactLevelFeatures(CuTexImage *list, unsigned int oldLen, float **newLevelFeatures, unsigned int newLen, TopKData &topKData, unsigned int offset)
{
  if(newLen <= 0)
  {
    *newLevelFeatures = NULL;
    return;
  }

  cudaMalloc((void**)(newLevelFeatures), newLen*4*sizeof(float));

  list->BindTexture(texDataF4);

  int blockWidth = TOPK_BLOCK_SIZE;
  dim3 grid((oldLen + blockWidth - 1) / blockWidth);
  dim3 block(blockWidth);

  CompactLevelFeatures_Kernel<<<grid, block>>>((float4 *)(*newLevelFeatures), (unsigned int *)topKData.keys, offset, oldLen);

  CheckErrorCUDA("TopKCompactLevelFeatures");
}

#endif // (GPU_HESSIAN || GPU_SIFT_MODIFIED) && TOP_K_SELECTION

#if defined GENERATE_FEATURE_LIST_USING_ATOMICS && (defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED)

void ProgramCU::DetectionDataInit(int **featureTexLen, int len)
{
  cudaMalloc((void**)(featureTexLen), len*sizeof(int));
  cudaMemset(*featureTexLen, 0, len*sizeof(int));

  CheckErrorCUDA("ProgramCU::DetectionDataInit");
}

void ProgramCU::DetectionDataDownload(int *dst, int *featureTexLen, int len)
{
  cudaMemcpy(dst, featureTexLen, len*sizeof(int), cudaMemcpyDeviceToHost);
  cudaMemset(featureTexLen, 0, len*sizeof(int));

  CheckErrorCUDA("ProgramCU::DetectionDataDownload");
}

void ProgramCU::DetectionDataFinish(int **featureTexLen)
{
  cudaFree(*featureTexLen);
  *featureTexLen = NULL;

  CheckErrorCUDA("ProgramCU::DetectionDataFinish");
}

#endif // GENERATE_FEATURE_LIST_USING_ATOMICS && (GPU_HESSIAN || GPU_SIFT_MODIFIED)


//////////////////////////////////////////////////////
void ProgramCU::FinishCUDA()
{
  cudaThreadSynchronize();
}

int ProgramCU::CheckErrorCUDA(const char* location)
{
  cudaError_t e = cudaGetLastError();

  if(e)
  {
    if(location)
      fprintf(stderr, "%s:\t",  location);
    fprintf(stderr, "%s\n",  cudaGetErrorString(e));
    // assert(0);
    return 1;
  }
  else {
    return 0; 
  }
}

void __global__ ConvertDOG_Kernel(float* d_result, int width, int height)
{
  int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
  int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;

  if((col < width) && (row < height))
  {
    int index = row * width  + col;
    float value = tex1Dfetch(texData, index);

    d_result[index] = ((col == 0) || (row == 0) || (col == width-1) || (row == height-1)) ? 0.5 : saturate(0.5+20.0*value);
  }
}

void ProgramCU::DisplayConvertDOG(CuTexImage* dog, CuTexImage* out)
{
  if(out->_cuData == NULL)
    return;

  int width = dog->GetImgWidth();
  int height = dog ->GetImgHeight();

  dog->BindTexture(texData);

  dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
  dim3 block(BLOCK_DIM, BLOCK_DIM);

  ConvertDOG_Kernel<<<grid, block>>>((float*) out->_cuData, width, height);
  ProgramCU::CheckErrorCUDA("DisplayConvertDOG");
}

void __global__ ConvertGRD_Kernel(float* d_result, int width, int height)
{
  int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
  int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;

  if((col < width) && (row < height))
  {
    int index = row * width  + col;
    float value = tex1Dfetch(texData, index << 1);

    d_result[index] = ((col == 0) || (row == 0) || (col == width-1) || (row == height-1)) ?	0.0 : saturate(5.0*value);
  }
}

void ProgramCU::DisplayConvertGRD(CuTexImage* got, CuTexImage* out)
{
  if(out->_cuData == NULL)
    return;

  int width = got->GetImgWidth();
  int height = got ->GetImgHeight();

  got->BindTexture(texData);

  dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
  dim3 block(BLOCK_DIM, BLOCK_DIM);

  ConvertGRD_Kernel<<<grid, block>>>((float*) out->_cuData, width, height);
  ProgramCU::CheckErrorCUDA("DisplayConvertGRD");
}

void __global__ ConvertKEY_Kernel(float4* d_result, int width, int height)
{
  int row = (blockIdx.y << BLOCK_LOG_DIM) + threadIdx.y;
  int col = (blockIdx.x << BLOCK_LOG_DIM) + threadIdx.x;

  if((col < width) && (row < height))
  {
    int index = row * width + col;
    float4 key = tex1Dfetch(texDataF4, index);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    // input: make_float4(result, dx, dy, ds);   =>  dx, dy, ds ... subpixel localizations (otherwise zero)
    //  result = response 16b half float | 14b unused | 2b type
    //  type = FEATURE_TYPE_DARK_BLOB    = 0
    //         FEATURE_TYPE_BRIGHT_BLOB  = 1
    //         FEATURE_TYPE_SADDLE_POINT = 2
    //         FEATURE_TYPE_NONE         = 3

    // extract feature type
    unsigned int typeValue = *((unsigned int *)(&key.x)); // __float_as_uint(key.x); // CUDA7.5
    typeValue = typeValue & 0x00000003u;

  //  int is_key = (typeValue != FEATURE_TYPE_NONE);
#else
    int is_key = ((key.x == 1.0f) || (key.x == -1.0f));
#endif // GPU_SIFT_MODIFIED || GPU_HESSIAN

    int inside = (col > 0) && (row > 0) && (row < height-1) && (col < width-1);
    float value = inside ? saturate(0.5 + 20.0 * tex1Dfetch(texData, index)) : 0.5;

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
    float4 result = make_float4(value, value, value, 0.0f);

    switch(typeValue)
    {
      case FEATURE_TYPE_DARK_BLOB:
        result = inside ? make_float4(1.0f, 0.0f, 0.0f, 1.0f) : result;
        break;
      case FEATURE_TYPE_BRIGHT_BLOB:
        result = inside ? make_float4(0.0f, 1.0f, 0.0f, 1.0f) : result;
        break;
      case FEATURE_TYPE_SADDLE_POINT:
        result = inside ? make_float4(0.0f, 0.0f, 1.0f, 1.0f) : result;
        break;
      case FEATURE_TYPE_NONE:
	  default:
        break;
    }

    d_result[index] = result;

    // if((typeValue != HESSIAN_FEATURE_TYPE_NONE) && inside)
    // {
    //   d_result[index-1] = result;
    //   d_result[index+1] = result;
    //   d_result[index-width] = result;
    //   d_result[index+width] = result;
    // }
#else
    d_result[index] = (is_key && inside) ? ((key.x > 0) ? make_float4(1.0f, 0.0f, 0.0f, 1.0f) : make_float4(0.0f, 1.0f, 0.0f, 1.0f)) : make_float4(value, value, value, 1.0f);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED
  }
}

void ProgramCU::DisplayConvertKEY(CuTexImage* key, CuTexImage* dog, CuTexImage* out)
{
  if(out->_cuData == NULL)
    return;

  int width = key->GetImgWidth();
  int height = key ->GetImgHeight();

  dog->BindTexture(texData);
  key->BindTexture(texDataF4);

  dim3 grid((width + BLOCK_DIM - 1)/ BLOCK_DIM,  (height + BLOCK_DIM - 1)/BLOCK_DIM);
  dim3 block(BLOCK_DIM, BLOCK_DIM);

  ConvertKEY_Kernel<<<grid, block>>>((float4*) out->_cuData, width, height);
}

void __global__ DisplayKeyPoint_Kernel(float4 * d_result, int num)
{
  int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;

  if(idx >= num)
    return;

  float4 key = tex1Dfetch(texDataF4, idx);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  // input in the feature list
  //  key.x: response 8b H | x 24b-14.10
  //  key.y: response 8b L | y 24b-14.10
  //  key.z: 2b type | 14b unused | scale 16b-8.8
  //  key.w: orientation

  // extract x position
  unsigned int tmpValue = *((unsigned int *)(&key.x)); // __float_as_uint(key.x); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.x = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

  // extract y position
  tmpValue = *((unsigned int *)(&key.y)); // __float_as_uint(key.y); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.y = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  d_result[idx] = make_float4(key.x, key.y, 0, 1.0f);
}

void ProgramCU::DisplayKeyPoint(CuTexImage* ftex, CuTexImage* out)
{
  int num = ftex->GetImgWidth();
  int block_width = 64;

  dim3 grid((num + block_width -1) /block_width);
  dim3 block(block_width);

  ftex->BindTexture(texDataF4);
  DisplayKeyPoint_Kernel<<<grid, block>>>((float4*) out->_cuData, num);

  ProgramCU::CheckErrorCUDA("DisplayKeyPoint");
}

void __global__ DisplayKeyBox_Kernel(float4* d_result, int num)
{
  int idx = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;

  if(idx >= num)
    return;

  int kidx = idx / 10;
  int vidx = idx - IMUL(kidx , 10);

  // fetch feature/keypoint
  float4 key = tex1Dfetch(texDataF4, kidx);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  // input in the feature list
  //  key.x: response 8b H | x 24b-14.10
  //  key.y: response 8b L | y 24b-14.10
  //  key.z: 2b type | 14 unused | scale 16b-8.8
  //  key.w: orientation

  // extract x position
  unsigned int tmpValue = *((unsigned int *)(&key.x)); // __float_as_uint(key.x); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.x = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

  // extract y position
  tmpValue = *((unsigned int *)(&key.y)); // __float_as_uint(key.y); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_POSITION_MASK;
  key.y = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_POSITION_PRECISION_BITS);

  // extract scale
  tmpValue = *((unsigned int *)(&key.z)); // __float_as_uint(key.z); // CUDA 7.5

  tmpValue = tmpValue & FIXED_POINT_SCALE_MASK;
  key.z = FIXED_POINT_TO_FLOAT(tmpValue, FIXED_POINT_SCALE_PRECISION_BITS);
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  float sz = fabs(key.z * 3.0f);
  ///////////////////////
  float s, c;
  __sincosf(key.w, &s, &c);
  ///////////////////////
  float dx = (vidx == 0) ? 0 : (((vidx <= 4) || (vidx >= 9)) ? sz : -sz);
  float dy = (vidx <= 1) ? 0 : (((vidx <= 2) || (vidx >= 7)) ? -sz : sz);

  float4 pos;

  pos.x = key.x + c * dx - s * dy;
  pos.y = key.y + c * dy + s * dx;
  pos.z = 0;
  pos.w = 1.0f;

  d_result[idx] = pos;
}

void ProgramCU::DisplayKeyBox(CuTexImage* ftex, CuTexImage* out)
{
  int len = ftex->GetImgWidth();
  int block_width = 32;

  dim3 grid((len * 10 + block_width -1) / block_width);
  dim3 block(block_width);

  ftex->BindTexture(texDataF4);

  DisplayKeyBox_Kernel<<<grid, block>>>((float4*) out->_cuData, len * 10);
}

///////////////////////////////////////////////////////////////////
inline void CuTexImage::BindTexture(textureReference& texRef)
{
  cudaBindTexture(NULL, &texRef, _cuData, &texRef.channelDesc, _numBytes);
}

inline void CuTexImage::BindTexture2D(textureReference& texRef)
{
#if defined(SIFTGPU_ENABLE_LINEAR_TEX2D)
  if ((_imgWidth*_numChannel*sizeof(float)) % 32)
    std::cout<<"Warning: Row length should be multiply of 32 !"<<std::endl;

  cudaBindTexture2D(0, &texRef, _cuData, &texRef.channelDesc, _imgWidth, _imgHeight, _imgWidth*_numChannel*sizeof(float));
#else
  cudaChannelFormatDesc desc;
  cudaGetChannelDesc(&desc, _cuData2D);
  cudaBindTextureToArray(&texRef, _cuData2D, &desc);
#endif
}

int ProgramCU::CheckCudaDevice(int device)
{
  int count = 0, device_used;

  if((cudaGetDeviceCount(&count) != cudaSuccess) || (count <= 0))
  {
    ProgramCU::CheckErrorCUDA("CheckCudaDevice");
    return 0;
  }
  else if(count == 1)
  {
    cudaDeviceProp deviceProp;

    if((cudaGetDeviceProperties(&deviceProp, 0) != cudaSuccess) || ((deviceProp.major == 9999) && (deviceProp.minor == 9999)))
    {
      fprintf(stderr, "CheckCudaDevice: no device supporting CUDA.\n");
      return 0;
    }
    else
    {
      GlobalUtil::_MemCapGPU = deviceProp.totalGlobalMem / 1024;
      GlobalUtil::_texMaxDimGL = 32768;

      if(GlobalUtil::_verbose) 
        fprintf(stdout, "NOTE: changing maximum texture dimension to %d\n", GlobalUtil::_texMaxDimGL);
    }
  }

  if((device > 0) && (device < count))  
  {
    cudaSetDevice(device);
    CheckErrorCUDA("cudaSetDevice\n");
  }

  cudaGetDevice(&device_used);

#if defined GPU_HESSIAN || defined GPU_SIFT_MODIFIED
  // we need CC 2.0 at least for feature list construction using atomics and topk selection
  cudaDeviceProp deviceProp;

  cudaGetDeviceProperties(&deviceProp, device_used);

  if(deviceProp.major < 2)
  {
    fprintf(stderr, "CheckCudaDevice: no device supporting CUDA CC 2.X or higher. Your device has just CC %d.%d.\n", deviceProp.major, deviceProp.minor);
    fprintf(stderr, "  Disable GENERATE_FEATURE_LIST_USING_ATOMICS and TOP_K_SELECTION in config.h and rebuild project.\n");
    return 0;
  }
#endif // GPU_HESSIAN || GPU_SIFT_MODIFIED

  if(device != device_used) 
    fprintf(stderr,  "\nERROR:   Cannot set device to %d\n"
        "\nWARNING: Use # %d device instead (out of %d)\n", device, device_used, count);
  return 1;
}

////////////////////////////////////////////////////////////////////////////////////////
// siftmatch funtions
//////////////////////////////////////////////////////////////////////////////////////////

#define MULT_TBLOCK_DIMX 128
#define MULT_TBLOCK_DIMY 1
#define MULT_BLOCK_DIMX (MULT_TBLOCK_DIMX)
#define MULT_BLOCK_DIMY (8 * MULT_TBLOCK_DIMY)


texture<uint4, 1, cudaReadModeElementType> texDes1;
texture<uint4, 1, cudaReadModeElementType> texDes2;

void __global__ MultiplyDescriptor_Kernel(int* d_result, int num1, int num2, int3* d_temp)
{
	int idx01 = (blockIdx.y  * MULT_BLOCK_DIMY),  idx02 = (blockIdx.x  * MULT_BLOCK_DIMX);

	int idx1 = idx01 + threadIdx.y, idx2 = idx02 + threadIdx.x;
	__shared__ int data1[17 * 2 * MULT_BLOCK_DIMY];
	int read_idx1 = idx01 * 8 +  threadIdx.x, read_idx2 = idx2 * 8;
	int col4 = threadIdx.x & 0x3, row4 = threadIdx.x >> 2;
	int cache_idx1 = IMUL(row4, 17) + (col4 << 2);

	///////////////////////////////////////////////////////////////
	//Load feature descriptors
	///////////////////////////////////////////////////////////////
#if MULT_BLOCK_DIMY == 16
	uint4 v = tex1Dfetch(texDes1, read_idx1);
	data1[cache_idx1]   = v.x;	data1[cache_idx1+1] = v.y;
	data1[cache_idx1+2] = v.z;	data1[cache_idx1+3] = v.w;
#elif MULT_BLOCK_DIMY == 8
	if(threadIdx.x < 64)
	{
		uint4 v = tex1Dfetch(texDes1, read_idx1);
		data1[cache_idx1]   = v.x;		data1[cache_idx1+1] = v.y;
		data1[cache_idx1+2] = v.z;		data1[cache_idx1+3] = v.w;
	}
#else
#error
#endif
	__syncthreads();

	///
	if(idx2 >= num2) return;
	///////////////////////////////////////////////////////////////////////////
	//compare descriptors

	int results[MULT_BLOCK_DIMY];
#pragma unroll
	for(int i = 0; i < MULT_BLOCK_DIMY; ++i) results[i] = 0;

#pragma unroll
	for(int i = 0; i < 8; ++i)
	{
		uint4 v = tex1Dfetch(texDes2, read_idx2 + i);
		unsigned char* p2 = (unsigned char*)(&v);
#pragma unroll
		for(int k = 0; k < MULT_BLOCK_DIMY; ++k)
		{
			unsigned char* p1 = (unsigned char*) (data1 + k * 34 + i *  4 + (i/4));
			results[k] += 	 ( IMUL(p1[0], p2[0])	+ IMUL(p1[1], p2[1])  
							 + IMUL(p1[2], p2[2])  	+ IMUL(p1[3], p2[3])  
							 + IMUL(p1[4], p2[4])  	+ IMUL(p1[5], p2[5])  
							 + IMUL(p1[6], p2[6])  	+ IMUL(p1[7], p2[7])  
							 + IMUL(p1[8], p2[8])  	+ IMUL(p1[9], p2[9])  
							 + IMUL(p1[10], p2[10])	+ IMUL(p1[11], p2[11])
							 + IMUL(p1[12], p2[12])	+ IMUL(p1[13], p2[13])
							 + IMUL(p1[14], p2[14])	+ IMUL(p1[15], p2[15]));
		}
	}

	int dst_idx = IMUL(idx1, num2)  + idx2;
	if(d_temp)
	{
		int3 cmp_result = make_int3(0, -1, 0);

#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1)
			{
				cmp_result = results[i] > cmp_result.x? 
				make_int3(results[i], idx1 + i, cmp_result.x) : 
				make_int3(cmp_result.x, cmp_result.y, max(cmp_result.z, results[i]));
				d_result[dst_idx + IMUL(i, num2)] = results[i];
			}
		}
		d_temp[ IMUL(blockIdx.y, num2) + idx2] = cmp_result; 
	}else
	{
#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1) d_result[dst_idx + IMUL(i, num2)] = results[i];
		}
	}

}


void ProgramCU::MultiplyDescriptor(CuTexImage* des1, CuTexImage* des2, CuTexImage* texDot, CuTexImage* texCRT)
{
	int num1 = des1->GetImgWidth() / 8;	
	int num2 = des2->GetImgWidth() / 8;
	dim3 grid(	(num2 + MULT_BLOCK_DIMX - 1)/ MULT_BLOCK_DIMX,
		(num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY);
	dim3 block(MULT_TBLOCK_DIMX, MULT_TBLOCK_DIMY);
	texDot->InitTexture( num2,num1);
	if(texCRT) texCRT->InitTexture(num2, (num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY, 32);
	des1->BindTexture(texDes1);
	des2->BindTexture(texDes2);

	MultiplyDescriptor_Kernel<<<grid, block>>>((int*)texDot->_cuData, num1, num2, 
												(texCRT? (int3*)texCRT->_cuData : NULL));
	ProgramCU::CheckErrorCUDA("MultiplyDescriptor");
}

texture<float, 1, cudaReadModeElementType> texLoc1;
texture<float2, 1, cudaReadModeElementType> texLoc2;
struct Matrix33{float mat[3][3];};



void __global__ MultiplyDescriptorG_Kernel(int* d_result, int num1, int num2, int3* d_temp,
										   Matrix33 H, float hdistmax, Matrix33 F, float fdistmax)
{
	int idx01 = (blockIdx.y  * MULT_BLOCK_DIMY);	
	int idx02 = (blockIdx.x  * MULT_BLOCK_DIMX);

	int idx1 = idx01 + threadIdx.y;	
	int idx2 = idx02 + threadIdx.x;
	__shared__ int data1[17 * 2 * MULT_BLOCK_DIMY];
	__shared__ float loc1[MULT_BLOCK_DIMY * 2];
	int read_idx1 = idx01 * 8 +  threadIdx.x ;
	int read_idx2 = idx2 * 8;
	int col4 = threadIdx.x & 0x3, row4 = threadIdx.x >> 2;
	int cache_idx1 = IMUL(row4, 17) + (col4 << 2);
#if MULT_BLOCK_DIMY == 16
	uint4 v = tex1Dfetch(texDes1, read_idx1);
	data1[cache_idx1]   = v.x;
	data1[cache_idx1+1] = v.y;
	data1[cache_idx1+2] = v.z;
	data1[cache_idx1+3] = v.w;
#elif MULT_BLOCK_DIMY == 8
	if(threadIdx.x < 64)
	{
		uint4 v = tex1Dfetch(texDes1, read_idx1);
		data1[cache_idx1]   = v.x;
		data1[cache_idx1+1] = v.y;
		data1[cache_idx1+2] = v.z;
		data1[cache_idx1+3] = v.w;
	}
#else
#error
#endif
	__syncthreads();
	if(threadIdx.x < MULT_BLOCK_DIMY * 2)
	{
		loc1[threadIdx.x] = tex1Dfetch(texLoc1, 2 * idx01 + threadIdx.x);
	}
	__syncthreads();
	if(idx2 >= num2) return;
	int results[MULT_BLOCK_DIMY];
	/////////////////////////////////////////////////////////////////////////////////////////////
	//geometric verification
	/////////////////////////////////////////////////////////////////////////////////////////////
	int good_count = 0;
	float2 loc2 = tex1Dfetch(texLoc2, idx2);
#pragma unroll
	for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
	{

		if(idx1 + i < num1)
		{
			float* loci = loc1 + i * 2;
			float locx = loci[0], locy = loci[1];
			//homography
			float x[3], diff[2];
			x[0] = H.mat[0][0] * locx + H.mat[0][1] * locy + H.mat[0][2];
			x[1] = H.mat[1][0] * locx + H.mat[1][1] * locy + H.mat[1][2];
			x[2] = H.mat[2][0] * locx + H.mat[2][1] * locy + H.mat[2][2];
			diff[0] = fabs(FDIV(x[0], x[2]) - loc2.x);
			diff[1] = fabs(FDIV(x[1], x[2]) - loc2.y);
			if(diff[0] < hdistmax && diff[1] < hdistmax)
			{
				//check fundamental matrix
				float fx1[3], ftx2[3], x2fx1, se; 
				fx1[0] = F.mat[0][0] * locx + F.mat[0][1] * locy + F.mat[0][2];
				fx1[1] = F.mat[1][0] * locx + F.mat[1][1] * locy + F.mat[1][2];
				fx1[2] = F.mat[2][0] * locx + F.mat[2][1] * locy + F.mat[2][2];

				ftx2[0] = F.mat[0][0] * loc2.x + F.mat[1][0] * loc2.y + F.mat[2][0];
				ftx2[1] = F.mat[0][1] * loc2.x + F.mat[1][1] * loc2.y + F.mat[2][1];
				//ftx2[2] = F.mat[0][2] * loc2.x + F.mat[1][2] * loc2.y + F.mat[2][2];

				x2fx1 = loc2.x * fx1[0]  + loc2.y * fx1[1] + fx1[2];
				se = FDIV(x2fx1 * x2fx1, fx1[0] * fx1[0] + fx1[1] * fx1[1] + ftx2[0] * ftx2[0] + ftx2[1] * ftx2[1]);
				results[i] = se < fdistmax? 0: -262144;
			}else
			{
				results[i] = -262144;
			}
		}else
		{
			results[i] = -262144;
		}
		good_count += (results[i] >=0);
	}
	/////////////////////////////////////////////////////////////////////////////////////////////
	///compare feature descriptors anyway
	/////////////////////////////////////////////////////////////////////////////////////////////
	if(good_count > 0)
	{
#pragma unroll
		for(int i = 0; i < 8; ++i)
		{
			uint4 v = tex1Dfetch(texDes2, read_idx2 + i);
			unsigned char* p2 = (unsigned char*)(&v);
#pragma unroll
			for(int k = 0; k < MULT_BLOCK_DIMY; ++k)
			{
				unsigned char* p1 = (unsigned char*) (data1 + k * 34 + i *  4 + (i/4));
				results[k] += 	 ( IMUL(p1[0], p2[0])	+ IMUL(p1[1], p2[1])  
								 + IMUL(p1[2], p2[2])  	+ IMUL(p1[3], p2[3])  
								 + IMUL(p1[4], p2[4])  	+ IMUL(p1[5], p2[5])  
								 + IMUL(p1[6], p2[6])  	+ IMUL(p1[7], p2[7])  
								 + IMUL(p1[8], p2[8])  	+ IMUL(p1[9], p2[9])  
								 + IMUL(p1[10], p2[10])	+ IMUL(p1[11], p2[11])
								 + IMUL(p1[12], p2[12])	+ IMUL(p1[13], p2[13])
								 + IMUL(p1[14], p2[14])	+ IMUL(p1[15], p2[15]));
			}
		}
	}
	int dst_idx = IMUL(idx1, num2)  + idx2;
	if(d_temp)
	{
		int3 cmp_result = make_int3(0, -1, 0);
#pragma unroll
		for(int i= 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1)
			{
				cmp_result = results[i] > cmp_result.x? 
				make_int3(results[i], idx1 + i, cmp_result.x) : 
				make_int3(cmp_result.x, cmp_result.y, max(cmp_result.z, results[i]));
				d_result[dst_idx + IMUL(i, num2)] = max(results[i], 0);
			}else
			{
				break;
			}
		}
		d_temp[ IMUL(blockIdx.y, num2) + idx2] = cmp_result; 
	}else
	{
#pragma unroll
		for(int i = 0; i < MULT_BLOCK_DIMY; ++i)
		{
			if(idx1 + i < num1) d_result[dst_idx + IMUL(i, num2)] = max(results[i], 0);
			else break;
		}
	}

}


void ProgramCU::MultiplyDescriptorG(CuTexImage* des1, CuTexImage* des2,
		CuTexImage* loc1, CuTexImage* loc2, CuTexImage* texDot, CuTexImage* texCRT,
		float H[3][3], float hdistmax, float F[3][3], float fdistmax)
{
	int num1 = des1->GetImgWidth() / 8;	
	int num2 = des2->GetImgWidth() / 8;
	Matrix33 MatF, MatH;
	//copy the matrix
	memcpy(MatF.mat, F, 9 * sizeof(float));
	memcpy(MatH.mat, H, 9 * sizeof(float));
	//thread blocks
	dim3 grid(	(num2 + MULT_BLOCK_DIMX - 1)/ MULT_BLOCK_DIMX,
		(num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY);
	dim3 block(MULT_TBLOCK_DIMX, MULT_TBLOCK_DIMY);
	//intermediate results
	texDot->InitTexture( num2,num1);
	if(texCRT) texCRT->InitTexture( num2, (num1 + MULT_BLOCK_DIMY - 1)/MULT_BLOCK_DIMY, 3);
	loc1->BindTexture(texLoc1);	
	loc2->BindTexture(texLoc2);
	des1->BindTexture(texDes1);	
	des2->BindTexture(texDes2);
	MultiplyDescriptorG_Kernel<<<grid, block>>>((int*)texDot->_cuData, num1, num2, 
												(texCRT? (int3*)texCRT->_cuData : NULL),
												MatH, hdistmax, MatF, fdistmax);
}


texture<int,  1, cudaReadModeElementType> texDOT;

#define ROWMATCH_BLOCK_WIDTH 32
#define ROWMATCH_BLOCK_HEIGHT 1

void __global__  RowMatch_Kernel(int*d_dot, int* d_result, int num2, float distmax, float ratiomax)
{
#if ROWMATCH_BLOCK_HEIGHT == 1
	__shared__ int dotmax[ROWMATCH_BLOCK_WIDTH];
	__shared__ int dotnxt[ROWMATCH_BLOCK_WIDTH];
	__shared__ int dotidx[ROWMATCH_BLOCK_WIDTH];
	int	row = blockIdx.y;
#else
	__shared__ int x_dotmax[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	__shared__ int x_dotnxt[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	__shared__ int x_dotidx[ROWMATCH_BLOCK_HEIGHT][ROWMATCH_BLOCK_WIDTH];
	int*	dotmax = x_dotmax[threadIdx.y];
	int*	dotnxt = x_dotnxt[threadIdx.y];
	int*	dotidx = x_dotidx[threadIdx.y];
	int row = IMUL(blockIdx.y, ROWMATCH_BLOCK_HEIGHT) + threadIdx.y;
#endif

	int base_address = IMUL(row , num2);
	int t_dotmax = 0, t_dotnxt = 0, t_dotidx = -1;
	for(int i = 0; i < num2; i += ROWMATCH_BLOCK_WIDTH)
	{
		if(threadIdx.x + i < num2)
		{
			int v = tex1Dfetch(texDOT, base_address + threadIdx.x + i);//d_dot[base_address + threadIdx.x + i];//
			bool test = v > t_dotmax;
			t_dotnxt = test? t_dotmax : max(t_dotnxt, v);
			t_dotidx = test? (threadIdx.x + i) : t_dotidx;
			t_dotmax = test? v: t_dotmax;
		}
		__syncthreads();
	}
	dotmax[threadIdx.x] = t_dotmax;
	dotnxt[threadIdx.x] = t_dotnxt;
	dotidx[threadIdx.x] = t_dotidx;
	__syncthreads();
	
#pragma unroll
	for(int step = ROWMATCH_BLOCK_WIDTH/2; step >0; step /= 2)
	{
		if(threadIdx.x < step)
		{
			int v1 = dotmax[threadIdx.x], v2 = dotmax[threadIdx.x + step];
			bool test =  v2 > v1;
			dotnxt[threadIdx.x] = test? max(v1, dotnxt[threadIdx.x + step]) :max(dotnxt[threadIdx.x], v2);
			dotidx[threadIdx.x] = test? dotidx[threadIdx.x + step] : dotidx[threadIdx.x];
			dotmax[threadIdx.x] = test? v2 : v1;
		}
		__syncthreads();
	}
	if(threadIdx.x == 0)
	{
		float dist =  acos(min(dotmax[0] * 0.000003814697265625f, 1.0));
		float distn = acos(min(dotnxt[0] * 0.000003814697265625f, 1.0));
		//float ratio = dist / distn;
		d_result[row] = (dist < distmax) && (dist < distn * ratiomax) ? dotidx[0] : -1;//?  : -1;
	}

}


void ProgramCU::GetRowMatch(CuTexImage* texDot, CuTexImage* texMatch, float distmax, float ratiomax)
{
	int num1 = texDot->GetImgHeight();
	int num2 = texDot->GetImgWidth();
	dim3 grid(1, num1/ROWMATCH_BLOCK_HEIGHT);
	dim3 block(ROWMATCH_BLOCK_WIDTH, ROWMATCH_BLOCK_HEIGHT);
	texDot->BindTexture(texDOT);
	RowMatch_Kernel<<<grid, block>>>((int*)texDot->_cuData,
		(int*)texMatch->_cuData, num2, distmax, ratiomax);
}

#define COLMATCH_BLOCK_WIDTH 32

//texture<int3,  1, cudaReadModeElementType> texCT;

void __global__  ColMatch_Kernel(int3*d_crt, int* d_result, int height, int num2, float distmax, float ratiomax)
{
	int col = COLMATCH_BLOCK_WIDTH * blockIdx.x + threadIdx.x;
	if(col >= num2) return;
	int3 result = d_crt[col];//tex1Dfetch(texCT, col);
	int read_idx = col + num2;
	for(int i = 1; i < height; ++i, read_idx += num2)
	{
		int3 temp = d_crt[read_idx];//tex1Dfetch(texCT, read_idx);
		result = result.x < temp.x?
			make_int3(temp.x, temp.y, max(result.x, temp.z)) :
			make_int3(result.x, result.y, max(result.z, temp.x));
	}

	float dist =  acos(min(result.x * 0.000003814697265625f, 1.0));
	float distn = acos(min(result.z * 0.000003814697265625f, 1.0));
		//float ratio = dist / distn;
	d_result[col] = (dist < distmax) && (dist < distn * ratiomax) ? result.y : -1;//?  : -1;

}

void ProgramCU::GetColMatch(CuTexImage* texCRT, CuTexImage* texMatch, float distmax, float ratiomax)
{
	int height = texCRT->GetImgHeight();
	int num2 = texCRT->GetImgWidth();
	//texCRT->BindTexture(texCT);
    dim3 grid((num2 + COLMATCH_BLOCK_WIDTH -1) / COLMATCH_BLOCK_WIDTH);
    dim3 block(COLMATCH_BLOCK_WIDTH);
	ColMatch_Kernel<<<grid, block>>>((int3*)texCRT->_cuData, (int*) texMatch->_cuData, height, num2, distmax, ratiomax);
}

#endif
